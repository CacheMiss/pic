#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <float.h>

#include "d_global_variables.h"

//*****************************************************************************
//  Function: injectWriteBlock
//  Purpose: Write a block's worth of particles back out to global memory
//  Parameters:
//  -------------------
//  partArray - The global particle array to write to
//  offset - The offset in the particle array to start at
//  posX[] - The x positions of the particles
//  posY[] - The y positions of the particles
//  velX[] - The x velocities of the particles
//  velY[] - The y velocities of the particles
//  velZ[] - the z velocities of the particles
//*****************************************************************************
__device__
void injectWriteBlock(float2 d_loc[], float3 d_vel[], 
                      int offset, 
                      volatile float s_posX[], volatile float s_posY[], 
                      volatile float s_velX[], volatile float s_velY[], volatile float s_velZ[]
)
{
   int particleStart = blockIdx.x * blockDim.x + offset + threadIdx.x;
   float2 loc;
   float3 vel;

   loc.x = s_posX[threadIdx.x];
   loc.y = s_posY[threadIdx.x];
   d_loc[particleStart] = loc;

   vel.x = s_velX[threadIdx.x];
   vel.y = s_velY[threadIdx.x];
   vel.z = s_velZ[threadIdx.x];
   d_vel[particleStart] = vel;
}

//*****************************************************************************
//  Function: inject
//  Purpose: Inject new particles into the top and bottom cells. Each particle
//           type has NIJ*NX1 new particles created
//  Caveats: This functino expects a block size of 512. To change this behavior
//           set the BLOCKSIZE variable to the desired block size
//  Parameters:
//  -------------------
//  eleHot[] - Two dimensional array represented as one dimension. Each row
//             is a separate particle. Particle values can be indexed as
//             follows:
//                [x*5 + 0] = Position x
//                [x*5 + 1] = Position y
//                [x*5 + 2] = Velocity x
//                [x*5 + 3] = Velocity y
//                [x*5 + 4] = Velocity z
//  eleCold[*5 + 5] - Two dimensional array represented as one dimension. Each row
//             is a separate particle. Particle values can be indexed as
//             follows:
//                [x*5 + 0] = Position x
//                [x*5 + 1] = Position y
//                [x*5 + 2] = Velocity x
//                [x*5 + 3] = Velocity y
//                [x*5 + 4] = Velocity z
//  ionHot[*5 + 5] - Two dimensional array represented as one dimension. Each row
//             is a separate particle. Particle values can be indexed as
//             follows:
//                [x*5 + 0] = Position x
//                [x*5 + 1] = Position y
//                [x*5 + 2] = Velocity x
//                [x*5 + 3] = Velocity y
//                [x*5 + 4] = Velocity z
//  ionCold[*5 + 5] - Two dimensional array represented as one dimension. Each row
//             is a separate particle. Particle values can be indexed as
//             follows:
//                [x*5 + 0] = Position x
//                [x*5 + 1] = Position y
//                [x*5 + 2] = Velocity x
//                [x*5 + 3] = Velocity y
//                [x*5 + 4] = Velocity z
//  DX - The delta between x positions in the grid
//  DY - The delta between y positions in the grid
//  numElectronsHot - The number of hot electrons
//  numElectronsCold - The number of cold electrons
//  numIonsHot - The number of hot ions
//  numIonsCold - The number of cold ions
//  randPool - An array of random numbers ranging from 0 to 1
//  randPoolSize - The number of elements in randPool
//  NX1 - The width of the grid
//  NY1 - The height of the grid
//  NIJ - Avg num particles per cell
//*****************************************************************************
__global__
void inject(float2 eleHotLoc[], float3 eleHotVel[], 
            float2 eleColdLoc[], float3 eleColdVel[],
            float2 ionHotLoc[], float3 ionHotVel[], 
            float2 ionColdLoc[], float3 ionColdVel[],
            const int botXStart, const int injectWidth,
            const float DX, const float DY,
            const int numElectronsHot, const int numElectronsCold, 
            const int numIonsHot, const int numIonsCold,
            float randPool[], const int randPoolSize,
            const unsigned int NX1, const unsigned int NY1,
            const unsigned int NIJ)
{
   const int RANDS_PER_THREAD = 24;
   int randOffset = blockIdx.x * blockDim.x * RANDS_PER_THREAD +
      threadIdx.x * RANDS_PER_THREAD;
   // An shared memory array for new particles
   extern __shared__ float sharedBeg[]; 
   volatile float *posX = sharedBeg;
   volatile float *posY = posX + blockDim.x;
   volatile float *velX = posY + blockDim.x;
   volatile float *velY = velX + blockDim.x;
   volatile float *velZ = velY + blockDim.x;
   // Check and make sure this thread has work, if it doesn't,
   // return here.
   bool hasWork = (blockIdx.x*blockDim.x+threadIdx.x < NIJ*NX1) ? true : false;
   const float velmass = static_cast<float>(1./D_RATO);
   float vpar;
   float tpar; 
   float stpar; // sin of tpar
   float ctpar; // cos of tpar
   //--------------------------------------------------------
   //                    electrons
   //--------------------------------------------------------
   //                     hot e
   //--------------------------------------------------------
   if(hasWork)
   {
      posX[threadIdx.x] = (float)(DX*NX1*randPool[randOffset]);
      posY[threadIdx.x] = (float)(DY*(NY1-1)+DY*randPool[randOffset+1]);
      vpar=(float)((1.414f*rsqrtf(D_SIGMA3))*
         sqrtf(-logf(1.0f-randPool[randOffset+2] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+3] - D_PI);
      // TODO - Figure out why vpar causes the "Double is not supported" warning
      velX[threadIdx.x] = (float)vpar*__sinf((float)tpar);
      vpar=(float)((1.414f*rsqrtf(D_SIGMA3))*
         sqrtf(-logf(1.0f-randPool[randOffset+4] + FLT_MIN)));
      tpar=(float)(D_TPI*randPool[randOffset+5] - D_PI);
      __sincosf(tpar, &stpar, &ctpar);
      velY[threadIdx.x] = vpar*stpar;
      velZ[threadIdx.x] = vpar*ctpar;
      posY[threadIdx.x] = posY[threadIdx.x]+D_DELT*velY[threadIdx.x];

      injectWriteBlock(eleHotLoc, eleHotVel, numElectronsHot, 
         posX, posY, velX, velY, velZ);
   }
   __syncthreads();

   //---------------------------------------------------------
   //                    cold e            
   //---------------------------------------------------------
   if(hasWork)
   {
      posX[threadIdx.x] = (float)(DX*injectWidth*randPool[randOffset+6]+botXStart);
      posY[threadIdx.x] = (float)(DY*randPool[randOffset+7]);
      vpar = (float)((1.414f*rsqrtf(D_SIGMA))*
         sqrtf(-logf(1-randPool[randOffset+8] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+9] - D_PI);
      velX[threadIdx.x] = (float)(vpar*__sinf(tpar));
      vpar = (float)((1.414f/sqrtf(D_SIGMA))*
         sqrtf(-logf(1-randPool[randOffset+10] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+11] - D_PI);
      __sincosf(tpar, &stpar, &ctpar);
      velY[threadIdx.x] = (float)(vpar*stpar);
      velY[threadIdx.x] = static_cast<float>(velY[threadIdx.x]+(1.1f*rsqrtf(D_SIGMA)));
      velZ[threadIdx.x] = vpar*ctpar;
      posY[threadIdx.x] = posY[threadIdx.x]+D_DELT*velY[threadIdx.x];

      posY[threadIdx.x] = max(posY[threadIdx.x], 0.0f);
      injectWriteBlock(eleColdLoc, eleColdVel, numElectronsCold, 
         posX, posY, velX, velY, velZ);
   }
   __syncthreads();

   //---------------------------------------------------------
   // hot ions
   //---------------------------------------------------------
   if(hasWork)
   {
      posX[threadIdx.x]= (float)(DX*NX1*randPool[randOffset+12]);
      posY[threadIdx.x]= (float)(DY*(NY1-1)+DY*randPool[randOffset+13]);
      vpar = (float)((1.414f*rsqrtf(velmass*D_SIGMA2))*
         sqrtf(-logf(1.0f-randPool[randOffset+14] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+15] - D_PI);
      velX[threadIdx.x] = (float)vpar*__sinf((float)tpar);
      vpar = (float)((1.414f*rsqrtf(velmass*D_SIGMA2))*
         sqrtf(-logf(1.0f-randPool[randOffset+16] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+17] - D_PI);
      __sincosf(tpar, &stpar, &ctpar);
      velY[threadIdx.x] = vpar*stpar;
      velY[threadIdx.x] = (float)(velY[threadIdx.x]-(1.5f*rsqrtf(D_SIGMA2*velmass)));
      velZ[threadIdx.x] = vpar*ctpar;
      posY[threadIdx.x] = posY[threadIdx.x]+D_DELT*velY[threadIdx.x];

      injectWriteBlock(ionHotLoc, ionHotVel, numIonsHot, 
         posX, posY, velX, velY, velZ);
   }
   __syncthreads();

   //-------------------------------------------------------
   //            cold ions          
   //-------------------------------------------------------
   if(hasWork)
   {
      posX[threadIdx.x] = (float)(DX*injectWidth*randPool[randOffset+6]+botXStart);
      posY[threadIdx.x] = (float)(DY*randPool[randOffset+19]);
      vpar = (float)((1.414f*rsqrtf(D_SIGMA1*velmass))*
         sqrtf(-logf(1.0f-randPool[randOffset+20] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+21] - D_PI);
      velX[threadIdx.x] = (float)vpar*__sinf((float)tpar);
      vpar = (float)((1.414f*rsqrtf(D_SIGMA1*velmass))*
         sqrtf(-logf(1.0f-randPool[randOffset+22] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+23] - D_PI);
      __sincosf(tpar, &stpar, &ctpar);
      velY[threadIdx.x] = vpar*stpar;
      velZ[threadIdx.x] = vpar*ctpar;
      posY[threadIdx.x] = posY[threadIdx.x]+D_DELT*velY[threadIdx.x];

      posY[threadIdx.x] = max(posY[threadIdx.x], 0.0f);
      injectWriteBlock(ionColdLoc, ionColdVel, numIonsCold, 
         posX, posY, velX, velY, velZ);
   }
}
