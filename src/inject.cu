#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <float.h>

#include "d_global_variables.h"
#include "device_utils.h"
#include "global_variables.h"
#include "inject.h"

//*****************************************************************************
//  Function: injectWriteBlock
//  Purpose: Write a block's worth of particles back out to global memory
//  Parameters:
//  -------------------
//  partArray - The global particle array to write to
//  offset - The offset in the particle array to start at
//  posX[] - The x positions of the particles
//  posY[] - The y positions of the particles
//  velX[] - The x velocities of the particles
//  velY[] - The y velocities of the particles
//  velZ[] - the z velocities of the particles
//*****************************************************************************
__device__
void injectWriteBlock(float2 d_loc[], float3 d_vel[], 
                      int offset, 
                      volatile float s_posX[], volatile float s_posY[], 
                      volatile float s_velX[], volatile float s_velY[], volatile float s_velZ[]
)
{
   int particleStart = blockIdx.x * blockDim.x + offset + threadIdx.x;
   float2 loc;
   float3 vel;

   loc.x = s_posX[threadIdx.x];
   loc.y = s_posY[threadIdx.x];
   d_loc[particleStart] = loc;

   vel.x = s_velX[threadIdx.x];
   vel.y = s_velY[threadIdx.x];
   vel.z = s_velZ[threadIdx.x];
   d_vel[particleStart] = vel;
}

//*****************************************************************************
//  Function: inject
//  Purpose: Inject new particles into the top and bottom cells. Each particle
//           type has NIJ*NX1 new particles created
//  Caveats: This functino expects a block size of 512. To change this behavior
//           set the BLOCKSIZE variable to the desired block size
//  Parameters:
//  -------------------
//  eleHot[] - Two dimensional array represented as one dimension. Each row
//             is a separate particle. Particle values can be indexed as
//             follows:
//                [x*5 + 0] = Position x
//                [x*5 + 1] = Position y
//                [x*5 + 2] = Velocity x
//                [x*5 + 3] = Velocity y
//                [x*5 + 4] = Velocity z
//  eleCold[*5 + 5] - Two dimensional array represented as one dimension. Each row
//             is a separate particle. Particle values can be indexed as
//             follows:
//                [x*5 + 0] = Position x
//                [x*5 + 1] = Position y
//                [x*5 + 2] = Velocity x
//                [x*5 + 3] = Velocity y
//                [x*5 + 4] = Velocity z
//  ionHot[*5 + 5] - Two dimensional array represented as one dimension. Each row
//             is a separate particle. Particle values can be indexed as
//             follows:
//                [x*5 + 0] = Position x
//                [x*5 + 1] = Position y
//                [x*5 + 2] = Velocity x
//                [x*5 + 3] = Velocity y
//                [x*5 + 4] = Velocity z
//  ionCold[*5 + 5] - Two dimensional array represented as one dimension. Each row
//             is a separate particle. Particle values can be indexed as
//             follows:
//                [x*5 + 0] = Position x
//                [x*5 + 1] = Position y
//                [x*5 + 2] = Velocity x
//                [x*5 + 3] = Velocity y
//                [x*5 + 4] = Velocity z
//  DX - The delta between x positions in the grid
//  DY - The delta between y positions in the grid
//  numElectronsHot - The number of hot electrons
//  numElectronsCold - The number of cold electrons
//  numIonsHot - The number of hot ions
//  numIonsCold - The number of cold ions
//  randPool - An array of random numbers ranging from 0 to 1
//  randPoolSize - The number of elements in randPool
//  NX1 - The width of the grid
//  NY1 - The height of the grid
//  numToInject - The number of particles to inject
//  numSecondaryCold - The number of secondary cold particles. The number of
//     primary cold particles is NIJ * NX1 = numSecondaryCold
//  SIGMA_HE - Hot Electron Sigma
//  SIGMA_HI - Hot Ion Sigma
//  SIGMA_CE - Cold Electron Sigma
//  SIGMA_CI - Cold Ion Sigma
//  SIGMA_HE_PERP - The perpendicular sigma for hot electrons (vx & vz)
//  SIGMA_HI_PERP - The perpendicular sigma for hot ions (vx & vz)
//  SIGMA_CE_SECONDARY - The sigma for the secondary cold electrons
//*****************************************************************************
__global__
void injectKernel(float2 eleHotLoc[], float3 eleHotVel[], 
            float2 eleColdLoc[], float3 eleColdVel[],
            float2 ionHotLoc[], float3 ionHotVel[], 
            float2 ionColdLoc[], float3 ionColdVel[],
            const int botXStart, const int injectWidth,
            const float DX, const float DY,
            const unsigned int numElectronsHot, const unsigned int numElectronsCold, 
            const unsigned int numIonsHot, const unsigned int numIonsCold,
            const float randPool[], const int randPoolSize,
            const unsigned int NX1, const unsigned int NY1,
            const unsigned int numToInject,
            const unsigned int numSecondaryCold,
            const float SIGMA_HE, const float SIGMA_HI,
            const float SIGMA_CE, const float SIGMA_CI,
            const float SIGMA_HE_PERP, const float SIGMA_HI_PERP,
            const float SIGMA_CE_SECONDARY)
{
   const int RANDS_PER_THREAD = 24;
   int randOffset = blockIdx.x * blockDim.x * RANDS_PER_THREAD +
      threadIdx.x * RANDS_PER_THREAD;
   // An shared memory array for new particles
   extern __shared__ float sharedBeg[]; 
   volatile float *posX = sharedBeg;
   volatile float *posY = posX + blockDim.x;
   volatile float *velX = posY + blockDim.x;
   volatile float *velY = velX + blockDim.x;
   volatile float *velZ = velY + blockDim.x;
   // Check and make sure this thread has work, if it doesn't,
   // return here.
   bool hasWork = (blockIdx.x*blockDim.x+threadIdx.x < numToInject) ? true : false;
   bool injectingSecondary = numToInject - (blockIdx.x*blockDim.x+threadIdx.x) <= numSecondaryCold;
   const float velmass = static_cast<float>(1./D_RATO);
   float vpar;
   float tpar; 
   float stpar; // sin of tpar
   float ctpar; // cos of tpar
   //--------------------------------------------------------
   //                    electrons
   //--------------------------------------------------------
   //                     hot e
   //--------------------------------------------------------
   if(hasWork)
   {
      // If SIGMA_HE_PERP is 0, use regular SIGMA_HE
      const float SIGMA_PERP = SIGMA_HE_PERP == 0 ? SIGMA_HE : SIGMA_HE_PERP;
      const float SIGMA_VERT = SIGMA_HE;
      posX[threadIdx.x] = (float)(DX*NX1*randPool[randOffset]);
      posY[threadIdx.x] = (float)(DY*(NY1-1)+DY*randPool[randOffset+1]);
      vpar=(float)((1.414f*rsqrtf(SIGMA_PERP))*
         sqrtf(-logf(1.0f-randPool[randOffset+2] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+3] - D_PI);
      velX[threadIdx.x] = (float)vpar*__sinf((float)tpar);
      // For sincos I need a range of -pi to pi
      tpar=(float)(D_TPI*randPool[randOffset+5] - D_PI);
      __sincosf(tpar, &stpar, &ctpar);
      vpar=(float)((1.414f*rsqrtf(SIGMA_VERT))*
         sqrtf(-logf(1.0f-randPool[randOffset+4] + FLT_MIN)));
      velY[threadIdx.x] = vpar*stpar - (1.1f*rsqrtf(SIGMA_VERT));
      vpar=(float)((1.414f*rsqrtf(SIGMA_PERP))*
         sqrtf(-logf(1.0f-randPool[randOffset+4] + FLT_MIN)));
      velZ[threadIdx.x] = vpar*ctpar;
      posY[threadIdx.x] = posY[threadIdx.x]+D_DELT*velY[threadIdx.x];

      injectWriteBlock(eleHotLoc, eleHotVel, numElectronsHot, 
         posX, posY, velX, velY, velZ);
   }
   __syncthreads();

   //---------------------------------------------------------
   //                    cold e            
   //---------------------------------------------------------
   if(hasWork)
   {
      posX[threadIdx.x] = (float)(DX*injectWidth*randPool[randOffset+6]+botXStart);
      posY[threadIdx.x] = (float)(DY*randPool[randOffset+7]);
      vpar = (float)((1.414f*rsqrtf(!injectingSecondary ? SIGMA_CE : SIGMA_CE_SECONDARY))*
         sqrtf(-logf(1-randPool[randOffset+8] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+9] - D_PI);
      velX[threadIdx.x] = (float)(vpar*__sinf(tpar));
      vpar = (float)((1.414f*rsqrtf(!injectingSecondary ? SIGMA_CE : SIGMA_CE_SECONDARY))*
         sqrtf(-logf(1-randPool[randOffset+10] + FLT_MIN)));
      // For sincos I need a range of -pi to pi
      tpar = (float)(D_TPI*randPool[randOffset+11] - D_PI);
      __sincosf(tpar, &stpar, &ctpar);
      velY[threadIdx.x] = (float)(vpar*stpar);
      velZ[threadIdx.x] = vpar*ctpar;
      posY[threadIdx.x] = posY[threadIdx.x]+D_DELT*velY[threadIdx.x];
      posY[threadIdx.x] = max(posY[threadIdx.x], 0.0f);

      injectWriteBlock(eleColdLoc, eleColdVel, numElectronsCold, 
         posX, posY, velX, velY, velZ);
   }
   __syncthreads();

   //---------------------------------------------------------
   // hot ions
   //---------------------------------------------------------
   if(hasWork)
   {
      // If SIGMA_HI_PERP is 0, use regular SIGMA_HE
      const float SIGMA_PERP = SIGMA_HI_PERP == 0 ? SIGMA_HI : SIGMA_HI_PERP;
      const float SIGMA_VERT = SIGMA_HI;
      posX[threadIdx.x]= (float)(DX*NX1*randPool[randOffset+12]);
      posY[threadIdx.x]= (float)(DY*(NY1-1)+DY*randPool[randOffset+13]);
      vpar = (float)((1.414f*rsqrtf(velmass*SIGMA_PERP))*
         sqrtf(-logf(1.0f-randPool[randOffset+14] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+15] - D_PI);
      velX[threadIdx.x] = (float)vpar*__sinf((float)tpar);
      // For sincos I need a range of -pi to pi
      tpar = (float)(D_TPI*randPool[randOffset+17] - D_PI);
      __sincosf(tpar, &stpar, &ctpar);
      vpar = (float)((1.414f*rsqrtf(velmass*SIGMA_VERT))*
         sqrtf(-logf(1.0f-randPool[randOffset+16] + FLT_MIN)));
      velY[threadIdx.x] = vpar*stpar;
      vpar = (float)((1.414f*rsqrtf(velmass*SIGMA_PERP))*
         sqrtf(-logf(1.0f-randPool[randOffset+16] + FLT_MIN)));
      velZ[threadIdx.x] = vpar*ctpar;
      posY[threadIdx.x] = posY[threadIdx.x]+D_DELT*velY[threadIdx.x];

      injectWriteBlock(ionHotLoc, ionHotVel, numIonsHot, 
         posX, posY, velX, velY, velZ);
   }
   __syncthreads();

   //-------------------------------------------------------
   //            cold ions          
   //-------------------------------------------------------
   if(hasWork)
   {
      posX[threadIdx.x] = (float)(DX*injectWidth*randPool[randOffset+6]+botXStart);
      posY[threadIdx.x] = (float)(DY*randPool[randOffset+19]);
      vpar = (float)((1.414f*rsqrtf(SIGMA_CI*velmass))*
         sqrtf(-logf(1.0f-randPool[randOffset+20] + FLT_MIN)));
      tpar = (float)(D_TPI*randPool[randOffset+21] - D_PI);
      velX[threadIdx.x] = (float)vpar*__sinf((float)tpar);
      vpar = (float)((1.414f*rsqrtf(SIGMA_CI*velmass))*
         sqrtf(-logf(1.0f-randPool[randOffset+22] + FLT_MIN)));
      // For sincos I need a range of -pi to pi
      tpar = (float)(D_TPI*randPool[randOffset+23] - D_PI);
      __sincosf(tpar, &stpar, &ctpar);
      velY[threadIdx.x] = vpar*stpar + (1.1f*rsqrtf(SIGMA_CI*velmass));
      velZ[threadIdx.x] = vpar*ctpar;
      posY[threadIdx.x] = posY[threadIdx.x]+D_DELT*velY[threadIdx.x];
      posY[threadIdx.x] = max(posY[threadIdx.x], 0.0f);

      injectWriteBlock(ionColdLoc, ionColdVel, numIonsCold, 
         posX, posY, velX, velY, velZ);
   }
}

void inject(DevMem<float2>& eleHotLoc, DevMem<float3>& eleHotVel, 
            DevMem<float2>& eleColdLoc, DevMem<float3>& eleColdVel,
            DevMem<float2>& ionHotLoc, DevMem<float3>& ionHotVel, 
            DevMem<float2>& ionColdLoc, DevMem<float3>& ionColdVel,
            const float DX, const float DY,
            unsigned int &numElectronsHot, unsigned int &numElectronsCold, 
            unsigned int &numIonsHot, unsigned int &numIonsCold,
				const unsigned int numToInject,
            const unsigned int numSecondaryCold,
            const DevMem<float>& randPool,
            const unsigned int NX1, const unsigned int NY1,
            const float SIGMA_HE, const float SIGMA_HI,
            const float SIGMA_CE, const float SIGMA_CI,
            const float SIGMA_HE_PERP, const float SIGMA_HI_PERP,
            const float SIGMA_CE_SECONDARY,
				const unsigned int injectWidth,
				const unsigned int injectStartX,
				DevStream &stream)
{
      const int injectThreadsPerBlock = MAX_THREADS_PER_BLOCK;
      dim3 injectNumBlocks(static_cast<unsigned int>(calcNumBlocks(injectThreadsPerBlock, numToInject)));
      dim3 injectBlockSize(injectThreadsPerBlock);
      int sharedMemoryBytes = sizeof(float) * 5 * injectThreadsPerBlock;
      stream.synchronize();
      checkForCudaError("RandomGPU");

      injectKernel<<<injectNumBlocks, injectBlockSize, sharedMemoryBytes, *stream>>>(
         eleHotLoc.getPtr(), eleHotVel.getPtr(), 
         eleColdLoc.getPtr(), eleColdVel.getPtr(), 
         ionHotLoc.getPtr(), ionHotVel.getPtr(), 
         ionColdLoc.getPtr(), ionColdVel.getPtr(), 
         injectStartX, injectWidth,
         DX, DY,
         numElectronsHot, numElectronsCold,
         numIonsHot, numIonsCold,
         randPool.getPtr(),
         static_cast<unsigned int>(randPool.size()),
         NX1, NY1, 
         numToInject, numSecondaryCold,
         SIGMA_HE, SIGMA_HI,
         SIGMA_CE, SIGMA_CI,
         SIGMA_HE_PERP, SIGMA_HI_PERP,
         SIGMA_CE_SECONDARY
         );
      checkForCudaError("Inject failed");

      numElectronsHot += numToInject;
      numElectronsCold += numToInject;
      numIonsHot += numToInject;
      numIonsCold += numToInject;
}
