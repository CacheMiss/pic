#include "hip/hip_runtime.h"
#include "potent2.h"
#include "dev_mem.h"
#include "dev_mem_reuse.h"
#include "device_utils.h"
#include "global_variables.h"
#include "particle_allocator.h"
#include "pic_utils.h"

#include <hipfft/hipfft.h>

#include <stdio.h>
#include <string>

//******************************************************************************
// Name: checkCufftStatus
// Code Type: Kernel
// Purpose: Check a cufft call for errors. If one is found, in all cases print
//          an error message. If in debug mode, throw an exception, otherwise
//          exit.
// Parameters:
// ----------------
// returnCode - The return code set by a cufft routine
//******************************************************************************
void checkCufftStatus(hipfftResult returnCode)
{
   std::string errorString;
   FILE *file;

   switch(returnCode)
   {
   case HIPFFT_INVALID_PLAN:
      errorString = "ERROR: CUFFT Invalid Plan";
      break;
   case HIPFFT_ALLOC_FAILED:
      errorString = "ERROR: CUFFT Alloc Failed";
      break;
   case HIPFFT_INVALID_TYPE:
      errorString = "ERROR: CUFFT Invalid Type";
      break;
   case HIPFFT_INVALID_VALUE:
      errorString = "ERROR: CUFFT Invalid Value";
      break;
   case HIPFFT_INTERNAL_ERROR:
      errorString = "ERROR: CUFFT Internal Error";
      break;
   case HIPFFT_EXEC_FAILED:
      errorString = "ERROR: CUFFT Exec Failed";
      break;
   case HIPFFT_SETUP_FAILED:
      errorString = "ERROR: CUFFT Setup Failed";
      break;
   case HIPFFT_INVALID_SIZE:
      errorString = "ERROR: CUFFT Invalid Size";
      break;
   };

   if(returnCode != HIPFFT_SUCCESS)
   {
      fprintf(stderr, "%s\n", errorString.c_str());
      file = fopen("errorLog.txt", "w");
      fprintf(file,"%s\n", errorString.c_str());
      fclose(file);
      assert(returnCode == HIPFFT_SUCCESS);
      exit(1);
   }
}

//******************************************************************************
// Name: initPb
// Code Type: Kernel
// Block Structure: One thread per grid column; Blocks and block size should
//                  be one dimensional
// Shared Memory Requirements: None
// Purpose: Initializes the pb array with voltage P0 and the imaginary
//          components to 0
// Parameters:
// ----------------
// pb - The array of size NX1 to initialize
// P0 - TBD
// size - The size of the array to convert
//******************************************************************************
__global__
void initPb(hipfftComplex pb[], const float P0, const unsigned int size)
{
   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   hipfftComplex val;
   val.x = P0;
   val.y = 0.0;

   if(threadX < size)
   {
      pb[threadX] = val;
   }
}

//******************************************************************************
// Name: loadHarmonics
// Code Type: Kernel
// Block Structure: One thread per grid column
// Shared Memory Requirements: None
// Purpose: Calculates the values for cokx
// Parameters:
// ----------------
// cokx - TBD
// size - Should be set to NX, which is 1 more than the number of elements
//******************************************************************************
__global__
void loadHarmonics(float cokx[], const unsigned int size)
{
   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   float num;

   if(threadX < size-1)
   {
      num = sin(D_PI * threadX / size);
      cokx[threadX]=4*(num*num);
   }
}

//******************************************************************************
// Name: calcPhif
// Code Type: Kernel
// Block Structure: One thread per grid column
// Shared Memory Requirements: None
// Purpose: Calculates the values for phif
// Parameters:
// ----------------
// phif - The phif array which will be set
// z - The z array is an intermediate working array and is expected to be of
//     size NX1 * NY1
// yyy - The yyy array is an intermediate working array and is expected to be of
//       size NX1 * NY1
// cokx - TBD size NX1
// pb - TBD size NX1
// c  - TBD size NX1 * NY1
// NX1 - The number of columns being calculated
// NY1 - The number of rows being calculated
// DX - The spacing between columns
// DY - The spacing between rows
//******************************************************************************
__global__
void calcPhif(hipfftComplex phif[],
              float z [],
              hipfftComplex yyy[],
              const float cokx[],
              const hipfftComplex pb[],
              const hipfftComplex c[],
              const unsigned int NX1,
              const unsigned int NY1,
              const float DX,
              const float DY
              )
{
   const unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   float2 f2zero;
   f2zero.x = 0;
   f2zero.y = 0;
   float2 tempYyy;
   float2 tempF2;
   float2 tempC;
   float tempZ;
   float tempCokx = cokx[threadX];
   const float dySquared = DY * DY;
   int index;
   int oneBack;
   int oneForward;

   if(threadX >= NX1)
   {
      return;
   }

   z[NX1 * (NY1-1) + threadX] = 0;
   yyy[NX1 * (NY1-1) + threadX] = pb[threadX];
   phif[threadX] = f2zero;

   for(int j = NY1 - 1; j  >= 1; j--)
   {
      index = NX1 * j + threadX;
      oneBack = index - NX1;
      tempZ = (float)1./((float)2. + (tempCokx*dySquared)-z[index]);
      z[oneBack] = tempZ;
      tempYyy = yyy[index];
      tempC = c[index];
      tempF2.x = tempZ * (tempYyy.x + dySquared * tempC.x);
      tempF2.y = tempZ * (tempYyy.y + dySquared * tempC.y);
      yyy[oneBack] = tempF2;
   }

   for(int j = 0; j < NY1; j++)
   {
      index = NX1 * j + threadX;
      oneForward = index + NX1;
      tempF2 = phif[index];
      tempZ = z[index];
      tempF2.x *= tempZ;
      tempF2.y *= tempZ;
      tempYyy = yyy[index];
      tempF2.x += tempYyy.x;
      tempF2.y += tempYyy.y;
      phif[oneForward] = tempF2;
   }
}

//******************************************************************************
// Name: complexToReal
// Code Type: Kernel
// Block Structure: One thread per value
// Shared Memory Requirements: blockDim.x * sizeof(float2)
// Purpose: Copies a hipfftComplex array into a real array. All complex values
//          are thrown away
// Parameters:
// ----------------
// complex - The hipfftComplex array
// real - The array of reals to be set
// size - The size of the array to convert
//******************************************************************************
__global__
void complexToReal(const hipfftComplex complex[], float real[], 
                   const unsigned int size)
{
   int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   extern __shared__ float2 temp[];

   if(threadX < size)
   {
      temp[threadIdx.x] = complex[threadX];
      real[threadX] = temp[threadIdx.x].x;
   }
}

//******************************************************************************
// Name: realToComplex
// Code Type: Kernel
// Block Structure: One thread per value
// Shared Memory Requirements: None
// Purpose: Copies a real array into a hipfftComplex array. All complex values
//          are set to 0
// Parameters:
// ----------------
// real - The array of reals
// complex - The hipfftComplex array to be set
// size - The size of the array to convert
//******************************************************************************
__global__
void realToComplex(const float real[], hipfftComplex complex[], 
                   const unsigned int size)
{
   int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   float2 temp;

   if(threadX < size)
   {
      temp.x = real[threadX];
      temp.y = 0;
      complex[threadX] = temp;
   }
}

//******************************************************************************
// Name: mapToPhi
// Code Type: Kernel
// Block Structure: Block size is 1 dimensional, Num blocks should be cover from
//                  0-NY vertically and 0-NX1 horizontally
// Shared Memory Requirements: None
// Purpose: Takes dens' internal memory layout for phi and maps it back to the
//          layout that field will expect. This process is grossly ineffecient
// Parameters:
// ----------------
// phi - The electric potential that will be passed to field
// packedFormat - The phi generated by dens
// packedWidth - The width of the packedFormat array
// packedHeight - The height of the packedFormat array
// phiWidth - The width of the phi array
//******************************************************************************
__global__
void mapToPhi(float phi[], const float packedFormat[],
              unsigned int packedWidth, unsigned int packedHeight,
              unsigned int phiWidth)
{
   int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   int threadY = blockDim.y * blockIdx.y + threadIdx.y;

   if(threadX >= packedWidth ||
      threadY >= packedHeight)
   {
      return;
   }

   phi[threadX * phiWidth + threadY] = 
      packedFormat[threadY * packedWidth + threadX];
}

//******************************************************************************
// Name: fixPhiSides
// Code Type: Kernel
// Block Structure: A single dimension of NY threads needed
// Shared Memory Requirements: None
// Purpose: Sets the periodic boundary condition for phi along the sides
// Parameters:
// ----------------
// phi - The electric potential
// width - The width of the phi array
// height - The height of the phi array
//******************************************************************************
__global__
void fixPhiSides(float phi[], 
                 unsigned int width,
                 unsigned int height
                 )
{
   int y = blockDim.x * blockIdx.x + threadIdx.x;
   
   phi[width * height + y] = phi[y];
}

//******************************************************************************
// Name: potent2
// Purpose: Calculate the electric potential at all of the grid points and
//          store it in phi.
// Input Parameters:
// -------------------
// dev_rho - The magnetic field at the grid points
//
// Output Parameters:
// -------------------
// dev_phi - The electrical potential at all of the grid points
//******************************************************************************
void potent2(DevMemF &dev_phi, const DevMemF &dev_rho)
{
   static bool first = true;

   unsigned int numThreads;
   dim3 blockSize;
   dim3 numBlocks;
   int sharedSize;
   DevMem<hipfftComplex, ParticleAllocator> dev_c(NX1 * NY);
   DevMem<hipfftComplex, ParticleAllocator> dev_pb(NX1);
   DevMem<float, DevMemReuse> dev_cokx(NX1);
   DevMem<hipfftComplex, ParticleAllocator> dev_phif(NY * NX1);
   DevMem<float, DevMemReuse> dev_z(NY1 * NX1);
   DevMem<hipfftComplex, ParticleAllocator> dev_yyy(NY1 * NX1);

   resizeDim3(blockSize, MAX_THREADS_PER_BLOCK / 2);
   resizeDim3(numBlocks, calcNumBlocks(256, NX1 * NY));
   hipDeviceSynchronize();
   checkForCudaError("Beginning of potent2");
   realToComplex<<<numBlocks, blockSize>>>(dev_rho.getPtr(), dev_c.getPtr(),
      dev_rho.size());
   hipDeviceSynchronize();
   checkForCudaError("realToComplex");
   static hipfftHandle rhoTransform;
   if(first)
   {
      checkCufftStatus(hipfftPlan1d(&rhoTransform, NX1, HIPFFT_C2C, NY));
   }
   checkCufftStatus(hipfftExecC2C(rhoTransform, dev_c.getPtr(), 
      dev_c.getPtr(), HIPFFT_FORWARD));
   //hipfftDestroy(rhoTransform);

   //ccccccccccccccccccccccccccccccccccccccccccccccccccccccc
   //           the poisson equation begins
   //ccccccccccccccccccccccccccccccccccccccccccccccccccccccc
   //     boundary conditions
   //ccccccccccccccccccccccccccccccccccccccccccccccccccccccc

   numThreads = MAX_THREADS_PER_BLOCK / 4;
   resizeDim3(blockSize, numThreads);
   resizeDim3(numBlocks, calcNumBlocks(numThreads, NX1));
   initPb<<<numBlocks, blockSize>>>(dev_pb.getPtr(), P0, dev_pb.size());
   checkForCudaError("initPb");

   static hipfftHandle pbTransform;
   if(first)
   {
      checkCufftStatus(hipfftPlan1d(&pbTransform, NX1, HIPFFT_C2C, 1));
   }
   hipDeviceSynchronize();
   checkForCudaError("Before cufft on dev_pb");
   checkCufftStatus(hipfftExecC2C(pbTransform, dev_pb.getPtr(), dev_pb.getPtr(),
      HIPFFT_FORWARD));
   //hipfftDestroy(pbTransform);

   // loading harmonics
   numThreads = MAX_THREADS_PER_BLOCK / 4;
   resizeDim3(blockSize, numThreads);
   resizeDim3(numBlocks, calcNumBlocks(numThreads, NX1));
   loadHarmonics<<<numBlocks, blockSize>>>(
      dev_cokx.getPtr(), dev_cokx.size()+1);
   checkForCudaError("loadHarmonics");

   numThreads = MAX_THREADS_PER_BLOCK / 8;
   resizeDim3(blockSize, numThreads);
   resizeDim3(numBlocks, calcNumBlocks(numThreads, NX1));
   hipDeviceSynchronize();
   checkForCudaError("Before calcPhif");
   calcPhif<<<numBlocks, numThreads>>>(dev_phif.getPtr(),
      dev_z.getPtr(), dev_yyy.getPtr(), dev_cokx.getPtr(), dev_pb.getPtr(),
      dev_c.getPtr(), NX1, NY1, DX, DY);
   checkForCudaError("calcPhif");

   static hipfftHandle phifTransform;
   if(first)
   {
      checkCufftStatus(hipfftPlan1d(&phifTransform, NX1, HIPFFT_C2C, NY));
   }
   hipDeviceSynchronize();
   checkForCudaError("Before inverse cufft on phif");
   hipfftExecC2C(phifTransform, dev_phif.getPtr(), 
      dev_phif.getPtr(), HIPFFT_BACKWARD);
   //hipfftDestroy(phifTransform);

   // Make space for transpose
   dev_yyy.freeMem();

   //DevMemF dev_tempPhi(NY * NX1);
   numThreads = MAX_THREADS_PER_BLOCK / 2;
   sharedSize = numThreads * sizeof(float2);
   resizeDim3(blockSize, numThreads);
   resizeDim3(numBlocks, calcNumBlocks(numThreads, NX1 * NY));
   hipDeviceSynchronize();
   checkForCudaError("Before final complex to real call in potent2");
   complexToReal<<<numBlocks, numThreads, sharedSize>>>(
      dev_phif.getPtr(), dev_phi.getPtr(), dev_phif.size());
   checkForCudaError("potent2::complexToReal");

   hipDeviceSynchronize();
   checkForCudaError("Before potent2 divVector");
   // Normalize the inverse transform
   divVector(dev_phi, float(NX1));
   checkForCudaError("potent2::divVector");

   first = true;
}
