#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014, Stephen C. Sewell
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.
// 
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
// ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
// ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
////////////////////////////////////////////////////////////////////////////////
#include "dens.h"

#include "array2d.h"
#include "dev_mem.h"
#include "dev_mem_reuse.h"
#include "device_utils.h"
#include "global_variables.h"
#include "particle_allocator.h"
#include "pic_utils.h"
#include "typedefs.h"

#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#ifndef NO_THRUST
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#endif

#ifdef DEVEMU
#include <vector>
#endif


struct ParticleBoundaries
{
   unsigned int beg;
   unsigned int end;

   ParticleBoundaries()
   {}

   ParticleBoundaries(unsigned int b, unsigned int e)
      :beg(b), end(e)
   { }
};


//******************************************************************************
// Function: findQuadIndices
// Code Type: Device
// Block Structure: 1 thread per element in rho to be considered
//                  (Normally NY x NX1)
// Purpose: Calculate the indices in global memory to access.  These are used 
//          to pull the proper values from bucketBeg and bucketEnd
// Input Parameters:
// -------------------
// threadX - The global x index of this thread
// threadY - The global y index of this thread
// bucketWidth - The number of buckets wide rho is
// left - True if the grid point has no elements to its left
// bottom - True if the grid point has no elements beneath it
//
// Output Parameters:
// -------------------
// gQuad1 - The index in the global beg and end array associated with this
//          threads quadrant 1
// gQuad2 - The index in the global beg and end array associated with this
//          threads quadrant 2
// gQuad3 - The index in the global beg and end array associated with this
//          threads quadrant 3
// gQuad4 - The index in the global beg and end array associated with this
//          threads quadrant 4
// quad1 - The index in the local beg and end array associated with this
//          threads quadrant 1
// quad2 - The index in the local beg and end array associated with this
//          threads quadrant 2
// quad3 - The index in the local beg and end array associated with this
//          threads quadrant 3
// quad4 - The index in the local beg and end array associated with this
//          threads quadrant 4
//******************************************************************************
__device__
void findQuadIndices(const unsigned int threadX, 
                     const unsigned int threadY,
                     const unsigned int bucketWidth,
                     const bool left, const bool bottom,
                     unsigned int &gQuad1, unsigned int &gQuad2,
                     unsigned int &gQuad3, unsigned int &gQuad4,
                     unsigned int &quad1, unsigned int &quad2,
                     unsigned int &quad3, unsigned int &quad4)
{
   // Calculate the indices in global memory to access
   // These are used to pull the proper values from bucketBeg and
   // bucketEnd
   gQuad1 = bucketWidth * threadY + threadX;
   if(!left)
   {
      gQuad2 = gQuad1 - 1;
   }
   // If there is nothing to the left, respect the periodic boundary
   // conditions and wrap to the other side of the grid
   else
   {
      gQuad2 = gQuad1 + bucketWidth-1;
   }
   gQuad3 = gQuad2 - bucketWidth;
   gQuad4 = gQuad1 - bucketWidth;

   // The index into the shared memory bucket beg and end variables
   // The + 1 values in this calculation exist because a border of
   // one bucket is needed around the left and bottom
   quad1 = (blockDim.x + 1) * (threadIdx.y + 1) + 
      threadIdx.x + 1;
   quad2 = quad1 - 1;
   quad3 = quad2 - (blockDim.x + 1);
   quad4 = quad1 - (blockDim.x + 1);
}

//******************************************************************************
// Function: densGridPointsLoadShared
// Code Type: Device
// Block Structure: 1 thread per element in rho to be considered
//                  (Normally NY x NX1)
// Purpose: Transfer area information from global memory to shared memory
//
// Input Parameters:
// -------------------
// area1 - Global memory to load a1 values from
// area2 - Global memory to load a2 values from
// area3 - Global memory to load a3 values from
// area4 - Global memory to load a4 values from
// gQuad1 - The index in the global beg and end array associated with this
//          threads quadrant 1
// gQuad2 - The index in the global beg and end array associated with this
//          threads quadrant 2
// gQuad3 - The index in the global beg and end array associated with this
//          threads quadrant 3
// gQuad4 - The index in the global beg and end array associated with this
//          threads quadrant 4
// quad1 - The index in the local beg and end array associated with this
//          threads quadrant 1
// quad2 - The index in the local beg and end array associated with this
//          threads quadrant 2
// quad3 - The index in the local beg and end array associated with this
//          threads quadrant 3
// quad4 - The index in the local beg and end array associated with this
//          threads quadrant 4
//
// Output Parameters:
// -------------------
// a1 - The a1 value associated with this grid point
// a2 - The a2 value associated with this grid point
// a3 - The a3 value associated with this grid point
// a4 - The a4 value associated with this grid point
//******************************************************************************
__device__
void densGridPointsLoadShared(const float* __restrict area1, 
                              const float* __restrict area2,
                              const float* __restrict area3, 
                              const float* __restrict area4,
                              float* __restrict a1, 
                              float* __restrict a2,
                              float* __restrict a3, 
                              float* __restrict a4,
                              const unsigned int quad1,
                              const unsigned int quad2,
                              const unsigned int quad3,
                              const unsigned int quad4,
                              const unsigned int gQuad1,
                              const unsigned int gQuad2,
                              const unsigned int gQuad3,
                              const unsigned int gQuad4,
                              const bool left,
                              const bool bottom
                              )
{
   //////////////////////////////////////////////////////////////////////////
   // NOTE: Commented out assignments of a1-a4 are left for completeness.
   //       They are however, not necessary because of the geometry involved
   //////////////////////////////////////////////////////////////////////////

   // Values associated with quadrants around a grid point
   // Q1 = a4
   // Q2 = a3
   // Q3 = a1
   // Q4 = a2
   
   // Load shared memory
   //a1[quad1] = area1[gQuad1];
   //a2[quad1] = area2[gQuad1];
   a3[quad1] = area3[gQuad1];
   a4[quad1] = area4[gQuad1];
   
   // Pull the extra bottom row into shared memory
   if(!bottom && threadIdx.y == 0)
   {
      a1[quad4] = area1[gQuad4];
      a2[quad4] = area2[gQuad4];
      //a3[quad4] = area3[gQuad4];
      //a4[quad4] = area4[gQuad4];
   }
   // Pull the extra left row into shared memory
   if(threadIdx.x == 0)
   {
      //a1[quad2] = area1[gQuad2];
      //a2[quad2] = area2[gQuad2];
      a3[quad2] = area3[gQuad2];
      //a4[quad2] = area4[gQuad2];
   }
   // Get the bottom left corner
   if(!bottom && threadIdx.x == 0 && threadIdx.y == 0)
   {
      a1[quad3] = area1[gQuad3];
      //a2[quad3] = area2[gQuad3];
      //a3[quad3] = area3[gQuad3];
      //a4[quad3] = area4[gQuad3];
   }
}

//******************************************************************************
// Function: densGridPoints
// Code Type: Kernel
// Block Structure: 1 thread per element in rho to be considered
//                  (Normally NX x NY)
// Shared Memory Requirements: 
//     2 * (blockDim.x + 1) * (blockDim.y + 1) * sizeof(unsigned int) +
//     4 * particlesToBuffer * sizeof(float) * blockDim.x
// Purpose: Find the charge density at grid points the results for each
//          point are then added to the current contents of rho.
// Input Parameters:
// -------------------
// bucketWidth - The number of buckets wide the rho is
// bucketHeight - The number of buckets high rho is
// area1 - Global memory containing the area1 data for each grid point in row order
// area2 - Global memory containing the area2 data for each grid point in row order
// area3 - Global memory containing the area3 data for each grid point in row order
// area4 - Global memory containing the area4 data for each grid point in row order
// cold - False if the particle array is composed of cold particles;
//        True otherwise
// particlesToBuffer - The number of particles per thread to buffer into shared
//        memory.
// NIJ - The avg particles per cell
//
// Output Parameters:
// -------------------
// rho[] - The charge array for the grid points
//******************************************************************************
__global__
void densGridPoints(float* __restrict__ rho,
                    const unsigned int bucketWidth,
                    const unsigned int bucketHeight,
                    const float* __restrict__ area1,
                    const float* __restrict__ area2,
                    const float* __restrict__ area3,
                    const float* __restrict__ area4,
                    const bool cold,
                    const unsigned int particlesToBuffer,
                    const unsigned int NIJ
                    )
{
   extern __shared__ char sharedBase[];
   // Calculate all of the shared memory offsets
   float *a1 = reinterpret_cast<float*>(sharedBase);
   float *a2 = &a1[2 * (blockDim.x + 1)];
   float *a3 = &a2[2 * (blockDim.x + 1)];
   float *a4 = &a3[2 * (blockDim.x + 1)];
   // End of all the shared memory offset calculation

   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int threadY = blockDim.y * blockIdx.y + threadIdx.y;

   float rhoLocal = 0;

   // Return if there is no work
   bool hasWork = true;
   if(threadY >= bucketHeight ||
      threadX >= bucketWidth)
   {
      hasWork = false;
   }

   // The global bucket id number associated with this thread
   unsigned int gQuad1;
   unsigned int gQuad2;
   unsigned int gQuad3;
   unsigned int gQuad4;
   unsigned int quad1;
   unsigned int quad2;
   unsigned int quad3;
   unsigned int quad4;
   bool bottom = false;
   bool left = false;

   if(blockIdx.y == 0 && threadIdx.y == 0)
   {
      bottom = true;
   }

   if(blockIdx.x == 0 && threadIdx.x == 0)
   {
      left = true;
   }

   // Initialize all the values for the quadrants associated with 
   // this grid point
   findQuadIndices(threadX, threadY, bucketWidth,
      left, bottom,
      gQuad1, gQuad2, gQuad3, gQuad4,
      quad1, quad2, quad3, quad4);

   if(hasWork)
   {
      // Load bucketBeg and bucketEnd into shared memory
      densGridPointsLoadShared(area1, area2, area3, area4,
         a1, a2, a3, a4,
         quad1, quad2, quad3, quad4,
         gQuad1, gQuad2, gQuad3, gQuad4,
         left, bottom);
   }

   // Make sure all threads have finished their load into shared memory
   __syncthreads();

   if(hasWork)
   {
      // Values associated with quadrants around a grid point
      // Q1 = a4
      // Q2 = a3
      // Q3 = a1
      // Q4 = a2

      // Quadrant 1
      //    | *
      // ---|---
      //    |
      rhoLocal += a4[quad1];

      // Quadrant 2
      //  * |
      // ---|---
      //    |
      rhoLocal += a3[quad2];

      if(!bottom)
      {
         // Quadrant 3
         //    |
         // ---|---
         //  * |
         rhoLocal += a1[quad3];

         // Quadrant 4
         //    |
         // ---|---
         //    | *
         rhoLocal += a2[quad4];
      }
   }

   if(hasWork)
   {
      if(cold)
      {
         // Scale the charge of the cold particles
         // This allows us to use our memory to simulate more hot particles
         //rhoLocal = rhoLocal * 10 / NIJ;
         rhoLocal = rhoLocal / NIJ;
      }
      else
      {
         //rhoLocal = rhoLocal / NIJ;
         rhoLocal = rhoLocal / (NIJ * 10);
      }
      rho[bucketWidth * threadY + threadX] += rhoLocal;
   }
}

struct DbgArea
{
   unsigned int threadX;
   unsigned int localBeg;
   unsigned int localEnd;
   uint2 globalBeg;
   uint2 globalEnd;
};

//******************************************************************************
// Function: sumArea
// Code Type: Kernel
// Block Structure: 1 thread per grid point (Normally NX1 x NY grid points)
//                  Blocks cannot contain grid points that are above or below
//                  one another
// Shared Memory Requirements: 
//     4 * sizeof(float) * blockSize * particlesToBuffer + 2 * sizeof(uint2);
// Purpose: Four area values are calculated for each particle in the calcA 
//          kernel. Once these areas are known, sumArea is called to find the
//          the sum of the areas for all particles in each cell.
// Input Parameters:
// -------------------
// bucketBeg[] - An array containing the first particle associated with each
//               grid bucket
// bucketEnd[] - An array containing the last particle associated with each
//               grid bucket
// maxMinArray[] - A book keeping array of global memory which contains 
//                 numGridBins of elements
// a1[] - The a1 values for each particle
// a2[] - The a2 values for each particle
// a3[] - The a3 values for each particle
// a4[] - The a4 values for each particle
// numGridBins - The number of grid bins in the simulation area 
//               This is usually NX1 * NY
// numParticles - The number of particles being considered
// bufferSize - The number of particles to buffer within shared memory at
//              a time. Care should be taken to not select a value that would
//              require more shared memory than is present on the device.
//
// Output Parameters:
// -------------------
// a1Sum[] - The array containing the sums for a1
// a2Sum[] - The array containing the sums for a2
// a3Sum[] - The array containing the sums for a3
// a4Sum[] - The array containing the sums for a4
//******************************************************************************
__global__
void sumArea(float* __restrict__ a1Sum, float* __restrict__ a2Sum, 
             float* __restrict__ a3Sum, float* __restrict__ a4Sum, 
             const unsigned int* __restrict__ bucketBeg, 
             const unsigned int* __restrict__ bucketEnd,
             uint2* __restrict__ maxMinArray,
             const float* __restrict__ a1, const float* __restrict__ a2, 
             const float* __restrict__ a3, const float* __restrict__ a4,
             unsigned int numGridBins, unsigned int numParticles,
             int bufferSize=4)
{
   // Begin shared memory declarations
   extern __shared__ float begShared[];
   float *partBufA1 = begShared;
   float *partBufA2 = &partBufA1[blockDim.x * bufferSize];
   float *partBufA3 = &partBufA2[blockDim.x * bufferSize];
   float *partBufA4 = &partBufA3[blockDim.x * bufferSize];
   uint2 *blockBeg = 
      reinterpret_cast<uint2*>(&partBufA4[blockDim.x * bufferSize]);
   uint2 *blockEnd = blockBeg + 1;
   // End shared memory declarations

   uint2 *globalMaxMin = &maxMinArray[blockIdx.x];

   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int beg;
   unsigned int end;

   // For gBeg and gEnd x is the first particle associated with a cell
   // and y is the last particle associated with the the cell (exclusive)

   // The first particle range necessary for this block
   uint2 gBeg;
   // The second particle range necessary for this block
   uint2 gEnd;

   unsigned int tmp1;
   unsigned int tmp2;

   // Local copies for area sums
   float localA1 = 0;
   float localA2 = 0;
   float localA3 = 0;
   float localA4 = 0;

   // Boolean flag marked true if the cell this thread is associated
   // with has particles in it
   bool hasWork = true;

   gBeg.x = 0;
   gBeg.y = 0;
   gEnd = gBeg;

   if(threadX < numGridBins)
   {
      beg = bucketBeg[threadX];
      end = bucketEnd[threadX];
   }
   else
   {
      hasWork = false;
      beg = 0;
      end = 0;
   }

   // Check if this set of areas has work
   if(beg == end)
   {
      hasWork = false;
   }

   // Store the range of particles needed by the first thread in the block
   if(threadIdx.x == 0)
   {
      blockBeg->x = beg;
      blockBeg->y = end;
   }
   // Store the range of particles needed by the last thread in the block
   else if(threadIdx.x == blockDim.x - 1)
   {
      blockEnd->x = beg;
      blockEnd->y = end;
   }

   __syncthreads();

   tmp1 = blockBeg->x;
   tmp2 = blockBeg->y;

   // If the first thread in the block has particles, create a copy for each
   // thread to limit memory contention
   if(tmp1 != tmp2)
   {
      gBeg.x = tmp1;
      gBeg.y = tmp2;
   }

   tmp1 = blockEnd->x;
   tmp2 = blockEnd->y;

   // If the last thread in the block has particles, create a copy for each
   // thread to limit memory contention
   if(tmp1 != tmp2)
   {
      gEnd.x = tmp1;
      gEnd.y = tmp2;
   }

   // If I am unsure if I have an particles in this block, I have to use atomic
   // functions to find out
   if(gBeg.x == gBeg.y || gEnd.x == gEnd.y)
   {
      if(beg != end)
      {
         atomicMin(&globalMaxMin->x, beg);
         atomicMax(&globalMaxMin->y, end);
      }
      __syncthreads();
      // Load the global value into a register to prevent serialized access
      if(threadIdx.x == 0)
      {
         *blockBeg = *globalMaxMin;
      }
      __syncthreads();
      //gBeg.x = blockBeg->x;
      //gEnd.y = blockBeg->y;
      gBeg = *blockBeg;
      gEnd = *blockBeg;
   }

   // Align gBeg and gEnd to allow for coalescing memory operations
   gBeg.x = (gBeg.x / 16) * 16;
   gEnd.y = ((gEnd.y + 15) / 16) * 16;

   // The starting index in the global array to load from for this block
   unsigned int gStartIndex = gBeg.x;
   // The global index this thread will load
   unsigned int gThreadIndex = gStartIndex + threadIdx.x;
   // The shared memory index this thread will load to
   unsigned int localIndex = threadIdx.x;

   unsigned int loopInit;
   unsigned int loopEnd;
   unsigned int lastLoadedIndex;

   // gStartIndex = the beginning of the current section to load
   // gEnd.y = one past the last particle to load
   while(gStartIndex < gEnd.y)
   {
      // Fill the buffer with particles
      localIndex = threadIdx.x;
      for(int i = 0; i < bufferSize; i++)
      {
         // Make sure there really is a particle to load, then load it
         if(gThreadIndex < gEnd.y)
         {
            partBufA1[localIndex] = a1[gThreadIndex];
            partBufA2[localIndex] = a2[gThreadIndex];
            partBufA3[localIndex] = a3[gThreadIndex];
            partBufA4[localIndex] = a4[gThreadIndex];
            localIndex += blockDim.x;
            gThreadIndex += blockDim.x;
         }
      }

      __syncthreads();

      // If buffer isn't full, mark the end as the last valid particle
      if(gStartIndex + blockDim.x >= gEnd.y)
      {
         lastLoadedIndex = gEnd.y;
      }
      // Buffer is full, last particle is end of buffer
      else
      {
         lastLoadedIndex = gStartIndex + blockDim.x * bufferSize;
      }

      if(gStartIndex < end && lastLoadedIndex > beg && hasWork)
      {
         loopEnd = min(end, lastLoadedIndex) - gStartIndex;
         loopInit = (gStartIndex >= beg) ? 0 : beg - gStartIndex;
         for(int i = loopInit; i < loopEnd; i++)
         {
            localA1 += partBufA1[i];
            localA2 += partBufA2[i];
            localA3 += partBufA3[i];
            localA4 += partBufA4[i];
         }
      }

      gStartIndex += blockDim.x * bufferSize;
      __syncthreads();
   }

   if(hasWork)
   {
      a1Sum[threadX] = localA1;
      a2Sum[threadX] = localA2;
      a3Sum[threadX] = localA3;
      a4Sum[threadX] = localA4;
   }
}

//******************************************************************************
// Name: loadParticleLocations
// Code Type: Kernel
// Block Structure: 1 thread per particle
// Purpose: Values and bins are used in calcA, and densGridPoints.
//          This routine loads those arrays with values from the particle
//          array.
// Input Parameters:
// ----------------
// d_loc - The location array to load from
// size - The number of particles
// NX  - The width of the grid + 1
// NX1 - The width of the grid
// NY -  The height of the grid
// coldEle - If the particles are cold electrons, this is set to true
//
// Output Parameters:
// ----------------
// locCopy - The array to copy location data into. If NULL, no data is copied.
// associatedBin - Array to store the bin number for each location in
//******************************************************************************
__global__
void loadParticleLocations(const float2* __restrict__ d_loc, 
                           float2* __restrict__ locCopy, 
                           unsigned int* __restrict__ associatedBin,
                           const unsigned int size,
                           const unsigned int NX,
                           const unsigned int NX1,
                           const unsigned int NY,
                           bool coldEle)
{
   int particleIndex = blockDim.x * blockIdx.x + threadIdx.x;
   int index = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int row;
   unsigned int column;
   float2 loc;
   if(index < size)
   {
      loc = d_loc[particleIndex];
      if(locCopy != NULL)
      {
         locCopy[index] = loc;
      }
      // for 2^126 <= y <= 2^128, __fdividef(x,y) delivers a result of zero,
      column = static_cast<unsigned int>(__fdividef(loc.x, D_DX));
      // for 2^126 <= y <= 2^128, __fdividef(x,y) delivers a result of zero,
      row = static_cast<unsigned int>(__fdividef(loc.y, D_DY));
      if(coldEle && row == NY)
      {
         row = 0;
      }
      if(column == NX1)
      {
         column = 0;
      }
      associatedBin[index] = row * NX1 + column;
   }
}

////////////////////////////////////////////////////////////////////////////////
/// @brief
/// Find the number of particles actually stored in a particle array
///
/// @pre
/// Needs (8 * numThreads+1) bytes of shared memory per block
///
/// @param[in] pos
/// The positions of the particles
/// @param[in] size
/// The number of particles to check
/// @param[in] oobValue
/// The y position a particle will have if its out of bounds
/// @param[out] firstOobIdx
/// The index of the first particle that is out of bounds. If no oob particles
/// are found, firstOobIdx = size
////////////////////////////////////////////////////////////////////////////////
__global__
void findFirstOob(const float2* __restrict__ pos,
                  const unsigned int size,
                  const float oobValue,
                  unsigned int *firstOobIdx)
{
   extern __shared__ float2 s_loc[];
   unsigned int threadX = blockIdx.x * blockDim.x + threadIdx.x;
   if(threadX < size)
   {
      s_loc[threadIdx.x] = pos[threadX];
      if(threadIdx.x + 1 == blockDim.x && threadX + 1 < size)
      {
         s_loc[threadIdx.x+1] = pos[threadX + 1];
      }
   }
   __syncthreads();
   if(threadX < size)
   {
      bool hasNext = threadX + 1 < size;
      if(hasNext)
      {
         if(s_loc[threadIdx.x+1].y == oobValue && s_loc[threadIdx.x].y != oobValue)
         {
            //printf("oob particle found at index %d\n", threadX+1);
            *firstOobIdx = threadX + 1;
         }
      }
   }
}

////////////////////////////////////////////////////////////////////////////////
/// @brief
/// Calculates a1/tota, a2/tota, a3/tota, and a4/tota for every particle listed
/// in particleLocations. These values are used in densElectron and densIon.
/// This kernel also finds the number of particles actually stored in the
/// particle array
///
/// @pre
/// Block Structure: 1 thread per particle
/// Needs (8 * numThreads+1) bytes of shared memory per block
///
/// @param[in] pos
///    xy pairs of all the particles to be calculated
/// @param[out] area1
///    The area calculation for quadrant 3
/// @param[out] area2
///    The area calculation for quadrant 4
/// @param[out] area3
///    The area calculation for quadrant 2
/// @param[out] area4
///    The area calculation for quadrant 1
/// @param[out] firstOobIdx
///    The index of the first particle which has a y value of oobValue
/// @param[in] oobValue
///    The y value a location will have if it is out of bounds
/// @param[in] numParticles
///    The number of particles
/// @param[in] NY
///    The height of the grid
/// @param[in] DX
///    The horizontal grid spacing
/// @param[in] DY
///    The vertical grid spacing
/// @param[in] coldElectrons
///    This should be set to true of the kernel is operating on cold electrons
///
////////////////////////////////////////////////////////////////////////////////
__global__
void calcA(const float2* __restrict__ pos,
           float* __restrict__ area1,
           float* __restrict__ area2,
           float* __restrict__ area3,
           float* __restrict__ area4,
           unsigned int *firstOobIdx,
           const float oobValue,
           const unsigned int numParticles,
           const unsigned int NY,
           const float DX,
           const float DY,
           const bool coldElectrons)
{
   extern __shared__ float2 s_loc[];
   int threadX = blockDim.x * blockIdx.x + threadIdx.x;

   if(threadX < numParticles)
   {
      s_loc[threadIdx.x] = pos[threadX];
      if(threadIdx.x + 1 == blockDim.x && threadX + 1 < numParticles)
      {
         s_loc[threadIdx.x+1] = pos[threadX + 1];
      }
   }
   __syncthreads();
   if(threadX < numParticles)
   {
      int2 gridIndex;
      // for 2^126 <= y <= 2^128, __fdividef(x,y) delivers a result of zero,
      gridIndex.x = __fdividef(s_loc[threadIdx.x].x, DX);
      // for 2^126 <= y <= 2^128, __fdividef(x,y) delivers a result of zero,
      gridIndex.y = __fdividef(s_loc[threadIdx.x].y, DX);
      if (coldElectrons && gridIndex.y == NY)
      {
         gridIndex.y = 0; 
      }

      // Find the distance from the associated grid point that the particle is
      float dela = s_loc[threadIdx.x].x - gridIndex.x * DX;
      float delb = s_loc[threadIdx.x].y - gridIndex.y * DY;
      float a1;
      float a2;
      float a3;
      float a4;
      float tota;


      // Calculate the areas for the particle
      a1 = dela * delb;
      a2 = DX * delb-a1;
      a3 = DY * dela-a1;
      tota = DX * DY;
      a4 = tota - (a1 + a2 + a3);

      // Write the areas back to global memory
      // for 2^126 <= y <= 2^128, __fdividef(x,y) delivers a result of zero,
      area1[threadX] = __fdividef(a1, tota);
      area2[threadX] = __fdividef(a2, tota);
      area3[threadX] = __fdividef(a3, tota);
      area4[threadX] = __fdividef(a4, tota);

      bool hasNext = threadX + 1 < numParticles;
      if(hasNext)
      {
         if(s_loc[threadIdx.x+1].y == oobValue && s_loc[threadIdx.x].y != oobValue)
         {
            //printf("oob particle found at index %d\n", threadX+1);
            *firstOobIdx = threadX + 1;
         }
      }
   }
}

//******************************************************************************
// Name: fixGridSides
// Code Type: Kernel
// Block Structure: 1 thread y position in grid
// Shared Memory Requirements: None
// Purpose: Enforces the periodic boundary condition on Rho, essentially wrapping
//          the rho array so that it is circular
// Parameters:
// ----------------
// rhoe - The rho array for electrons
// rhoi - The rho array for ions
// logicalX - The width of the area that is calculated
// logicalY - The height of the area that is calculated
// physicalX - The number of columns allocated for the array
// physicalY - The number of rows allocated for the array
//******************************************************************************
__global__
void fixRhoGridSides(float* __restrict__ rhoe, float* __restrict__ rhoi,
                     const unsigned int logicalX, 
                     const unsigned int logicalY,
                     const unsigned int physicalX, 
                     const unsigned int physicalY
                     )
{
   float sum;
   int indexBottom = blockDim.x * blockIdx.x + threadIdx.x;
   int indexTop = indexBottom + physicalY * (logicalX - 1);

   if(indexBottom >= logicalY)
   {
      return;
   }

   sum = rhoe[indexBottom] + rhoe[indexTop];
   rhoe[indexBottom] = sum;
   rhoe[indexTop] = sum;
   sum = rhoi[indexBottom] + rhoi[indexTop];
   rhoi[indexBottom] = sum;
   rhoi[indexTop] = sum;
}

//******************************************************************************
// Name: fixRhoGridTopBottom
// Code Type: Kernel
// Block Structure: 1 thread x position in grid
// Shared Memory Requirements: None
// Purpose: Doubles the rho at the top and bottom of the grid
// Parameters:
// ----------------
// rhoe - The rho array for electrons
// rhoi - The rho array for ions
// width - The width of the area that is calculated
// height - The height of the area that is calculated
//******************************************************************************
__global__
void fixRhoGridTopBottom(float* __restrict__ rhoe, float* __restrict__ rhoi,
                         const unsigned int width, 
                         const unsigned int height
                         )
{
   unsigned int threadNum = blockDim.x * blockIdx.x + threadIdx.x;
   unsigned int distanceToTop = width * (height - 1);
   if(threadNum >= width)
   {
      return;
   }

   rhoe[threadNum] *= 2;
   rhoe[threadNum + distanceToTop] *= 2;
   rhoi[threadNum] *= 2;
   rhoi[threadNum + distanceToTop] *= 2;
}

//******************************************************************************
// Name: getFinalRho
// Code Type: Kernel
// Block Structure: 1 thread per grid point to be considered 
//                  Should be (X_GRD * Y_GRD threads)
// Shared Memory Requirements: None
// Purpose: Subtracts rhoe from rhoi
// Parameters:
// ----------------
// rhoe - The rho array for electrons
// rhoi - The rho array for ions
// size - The total number of grid points
//******************************************************************************
__global__ 
void getFinalRho(float* __restrict__ rho,
                 const float* __restrict__ rhoi,
                 const float* __restrict__ rhoe,
                 const unsigned int size)
{
   unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

   // Return if there is no work
   if(index >= size)
   {
      return;
   }

   rho[index] = rhoi[index] - rhoe[index];
}

__global__
void findUpperLowerBound(const unsigned int* __restrict__ binList,
                         unsigned int* __restrict__ beg, 
                         unsigned int* __restrict__ end,
                         unsigned int numParticles, 
                         unsigned int numBins)
{
   extern __shared__ char bytes[];
   unsigned int *scratch = reinterpret_cast<unsigned int *>(&bytes[0]);
   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   int index = threadIdx.x + 1;

   // If there is work to do
   if(threadX < numParticles)
   {
      // Grab the particle to the left of the block if there is one
      if(blockIdx.x != 0 && threadIdx.x == 0)
      {
         scratch[threadIdx.x] = binList[threadX - 1];
      }
      // Grab a particle
      scratch[index] = binList[threadX];
   }

   // Make sure all shared memory gets loaded
   __syncthreads();

   if(threadX < numParticles)
   {
      // Any bin boundary is both the end of one bin
      // and the start of the next
      if((blockIdx.x > 0 || threadIdx.x != 0) &&
         scratch[index] > scratch[threadIdx.x])
      {
         beg[scratch[index]] = threadX;
         end[scratch[threadIdx.x]] = threadX;
      }

      if(threadX + 1 == numParticles)
      {
         end[scratch[index]] = threadX + 1;
      }
   }
}

template<class BinListAllocator, class BegEndAllocator>
void findBounds(const DevMem<unsigned int, BinListAllocator> &dev_binList,
                DevMem<unsigned int, BegEndAllocator> &dev_beg, 
                DevMem<unsigned int, BegEndAllocator> &dev_end,
                unsigned int numParticles, unsigned int numBins, 
                hipStream_t stream
                )
{
   int  threadsInBlock = MAX_THREADS_PER_BLOCK / 2;
   dim3 blockSize(threadsInBlock);
   dim3 numBlocks(calcNumBlocks(threadsInBlock, numParticles));
   int  sharedMemoryBytes = sizeof(unsigned int) * (threadsInBlock + 1);

   findUpperLowerBound<<<numBlocks, blockSize, sharedMemoryBytes, stream>>>(
      dev_binList.getPtr(), dev_beg.getPtr(), dev_end.getPtr(), 
      numParticles, numBins);
   checkForCudaError("findUpperLowerBound");
}

//******************************************************************************
// Name: calcIntermediateRho
// Purpose: Calculates rho accross the grid for a particle array and adds it
//          to the contents in dev_rho
// Parameters:
// ----------------
// dev_rho - A pointer to device memory used to store the results in
// d_partLoc - The location array for the particles
// d_partVel - The velocity array for the particles
// numParticles - The number of particles stored in dev_particles
// cold - True if the particles are cold
// electron - True of the particles are electrons
// sortParticleArray - Don't make a temporary copy of particle locations.
//                     Sort the entire particle array.
// stream - Stream for processing in calcIntermediateRho
//******************************************************************************
void calcIntermediateRho(DevMemF &dev_rho,
                         DevMem<float2> &d_partLoc, 
                         DevMem<float3> &d_partVel,
                         unsigned int& numParticles,
                         bool cold,
                         bool electron,
                         bool sortParticleArray,
                         hipStream_t stream)
{
   DeviceStats &dev(DeviceStats::getRef());
   DevMem<float2, ParticleAllocator> dev_particleLocations(0);
   if(!sortParticleArray)
   {
      dev_particleLocations.resize(numParticles);
   }
   // A vector which maps grid buckets back to particle indices
   DevMem<unsigned int, ParticleAllocator> dev_gridBuckets(numParticles);
   DevMem<unsigned int, DevMemReuse> dev_bucketBegin(NY * NX1);
   DevMem<unsigned int, DevMemReuse> dev_bucketEnd(NY * NX1);
   dev_bucketBegin.zeroMem();
   dev_bucketEnd.zeroMem();
   dim3 numBlocks;
   dim3 blockSize;
   int threadsInBlock;
   int threadsX;
   int threadsY;
   std::size_t sharedMemoryBytes;
   unsigned int particlesToBuffer = 4;

   // Sort particles before calling densGridPoints

   // Run loadParticleLocations to generate copies of all particle
   // locations and calculate the bin associated with each of them.
   // This bin is then used to sort the particles
   threadsInBlock = dev.maxThreadsPerBlock / 2;
   blockSize = dim3(threadsInBlock);
   numBlocks = dim3(static_cast<unsigned int>(calcNumBlocks(threadsInBlock, numParticles)));
   hipStreamSynchronize(stream);
   checkForCudaError("Before loadParticleLocations");
   loadParticleLocations<<<numBlocks, blockSize, 0, stream>>>(
      d_partLoc.getPtr(),
      dev_particleLocations.getPtrUnsafe(),
      dev_gridBuckets.getPtr(),
      numParticles,
      NX, NX1, NY,
      cold && electron);
   hipStreamSynchronize(stream);
   checkForCudaError("densLoadSortArrays failed");

   // DEBUG
   //std::vector<int> h_keys(100);
   //std::vector<float2> h_val(100);
   //for(int i = 0; i < 100; i++)
   //{
   //   h_keys[i] = 100-i;
   //   h_val[i].x = i;
   //   h_val[i].y = 100-i;
   //   //h_val[i] = 100-i;
   //}
   //DevMem<int> dev_keys(100);
   //dev_keys = h_keys;
   //DevMem<float2> dev_val(100);
   //dev_val = h_val;
   //picSort(dev_keys, dev_val);
   //checkForCudaErrorSync("Sort test");
   //dev_keys.copyArrayToHost(&h_keys[0]);
   //dev_val.copyArrayToHost(&h_val[0]);
   // END DEBUG

   // Once all particles are loaded and have buckets; sort 
   // the buckets so that I can find all particles within a 
   // certain bucket
   if(!sortParticleArray)
   {
      assert(dev_gridBuckets.size() == dev_particleLocations.size());
      picSort(dev_gridBuckets, dev_particleLocations);
   }
   else
   {
      // Its faster to sort twice then to move positions and velocities
      // around together. Copy the key data for the second sort
      DevMem<unsigned int, ParticleAllocator> tmpBuckets(dev_gridBuckets);
      picSort(dev_gridBuckets, d_partLoc);
      picSort(tmpBuckets, d_partVel);
   }
   checkForCudaError("Before findBounds");

   /*
   // Begin DEBUG
   hipStreamSynchronize(stream);
   std::vector<unsigned int> h_gridBuckets(dev_gridBuckets.size());
   std::vector<float2> h_particleLocations(dev_particleLocations.size());
   dev_gridBuckets.copyArrayToHost(&h_gridBuckets[0]);
   dev_particleLocations.copyArrayToHost(&h_particleLocations[0]);
   FILE *dbgSort = fopen("dbgSort.txt", "w");
   for(int i = 0; i < h_gridBuckets.size(); i++)
   {
      fprintf(dbgSort, "%u %f %f\n", h_gridBuckets[i], h_particleLocations[i].x, h_particleLocations[i].y);
   }
   fclose(dbgSort);
   // End DEBUG
   */
   
   // find the beginning and end of each bucket's list of points
   findBounds(dev_gridBuckets, 
      dev_bucketBegin, 
      dev_bucketEnd,
      numParticles, 
      static_cast<unsigned int>(dev_bucketBegin.size()), 
      stream);

   /*
   // Begin DEBUG
   hipStreamSynchronize(stream);
   std::vector<unsigned int> h_bucketBegin(dev_bucketBegin.size());
   std::vector<unsigned int> h_bucketEnd(dev_bucketEnd.size());
   dev_bucketBegin.copyArrayToHost(&h_bucketBegin[0]);
   dev_bucketEnd.copyArrayToHost(&h_bucketEnd[0]);
   FILE *dbgBounds = fopen("dbgBounds.txt", "w");
   for(int i = 0; i < h_bucketBegin.size(); i++)
   {
      fprintf(dbgBounds, "%u %u\n", h_bucketBegin[i], h_bucketEnd[i]);
   }
   fclose(dbgBounds);
   // End DEBUG
   */

   // Sum area enforces coalesced memory loads and stores, but to do
   // this it can sometimes read more data than it needs. The extra
   // 31 elements in each array exist to ensure that no uninitialized
   // memory is read
   DevMem<float, ParticleAllocator> dev_a1(numParticles + dev.warpSize-1);
   DevMem<float, ParticleAllocator> dev_a2(numParticles + dev.warpSize-1);
   DevMem<float, ParticleAllocator> dev_a3(numParticles + dev.warpSize-1);
   DevMem<float, ParticleAllocator> dev_a4(numParticles + dev.warpSize-1);
   DevMem<float, DevMemReuse> dev_a1Sum(NX1 * NY);
   DevMem<float, DevMemReuse> dev_a2Sum(NX1 * NY);
   DevMem<float, DevMemReuse> dev_a3Sum(NX1 * NY);
   DevMem<float, DevMemReuse> dev_a4Sum(NX1 * NY);

   dev_a1Sum.zeroMem();
   dev_a2Sum.zeroMem();
   dev_a3Sum.zeroMem();
   dev_a4Sum.zeroMem();
   
   // Calculate a1, a2, a3, and a4 for the now sorted particles
   threadsInBlock = 512;
   blockSize = dim3(threadsInBlock);
   numBlocks = dim3(static_cast<unsigned int>(calcNumBlocks(threadsInBlock, numParticles)));
   sharedMemoryBytes = d_partLoc.getElementSize() * (threadsInBlock + 1);
   float2 *locToUse;
   if(!sortParticleArray)
   {
      locToUse = dev_particleLocations.getPtr();
   }
   else
   {
      locToUse = d_partLoc.getPtr();
   }
   static HostMem<unsigned int> numRemainingParticles(1);
   numRemainingParticles[0] = numParticles;
   DevMem<unsigned int, DevMemReuse> d_firstOob = numRemainingParticles;
   calcA<<<numBlocks, blockSize, sharedMemoryBytes, stream>>>(
      locToUse,
      dev_a1.getPtr(),
      dev_a2.getPtr(),
      dev_a3.getPtr(),
      dev_a4.getPtr(),
      d_firstOob.getPtr(),
      OOB_PARTICLE,
      numParticles, 
      NY, DX, DY,
      cold && electron
      );
   checkForCudaError("calcA failed");
   checkCuda(hipStreamSynchronize(stream));
   numRemainingParticles = d_firstOob;
   // This value eventually gets set as the active number of particles in main
   // If the whole particle array was just sorted, it also becomes the total particle count
   numParticles = numRemainingParticles[0];


   //threadsInBlock = MAX_THREADS_PER_BLOCK / 8;
   threadsInBlock = 64;
   //threadsInBlock = dev.maxThreadsPerBlock / 8;
   blockSize = dim3(threadsInBlock);
   numBlocks = dim3(static_cast<unsigned int>(calcNumBlocks(threadsInBlock, NY * NX1)));
   sharedMemoryBytes = 
      4 * sizeof(float) * threadsInBlock * particlesToBuffer + 
      2 * sizeof(uint2);
   uint2 tmpVal;
   tmpVal.x = numParticles;
   tmpVal.y = 0;
#ifndef NO_THRUST
   DevMem<uint2, ParticleAllocator> dev_maxMinArray(numBlocks.x, tmpVal);
#else
   DevMem<uint2, ParticleAllocator> dev_maxMinArray(numBlocks.x);
   setDeviceArray(dev_maxMinArray.getPtr(), dev_maxMinArray.size(), tmpVal);
#endif
   checkForCudaError("Finished prep for sumArea");
   // For each bin in the grid, sum all the a1-a4 values.
   // Once this is complete, the charge at a point will just be a1+a2+a3+a4.
   sumArea<<<numBlocks, blockSize, sharedMemoryBytes, stream>>>(
      dev_a1Sum.getPtr(), dev_a2Sum.getPtr(), 
      dev_a3Sum.getPtr(), dev_a4Sum.getPtr(),
      dev_bucketBegin.getPtr(), dev_bucketEnd.getPtr(),
      dev_maxMinArray.getPtr(),
      dev_a1.getPtr(), dev_a2.getPtr(), 
      dev_a3.getPtr(), dev_a4.getPtr(),
      NY * NX1, numParticles, particlesToBuffer
      );

   checkCuda(hipStreamSynchronize(stream));
   checkForCudaError("sumArea");
   dev_maxMinArray.freeMem();
   dev_a1.freeMem();
   dev_a2.freeMem();
   dev_a3.freeMem();
   dev_a4.freeMem();

   // Calculate the particles effect on rho
   threadsX = MAX_THREADS_PER_BLOCK / 4;
   threadsY = 1;
   blockSize = dim3(threadsX, threadsY);
   numBlocks = dim3(static_cast<unsigned int>(calcNumBlocks(threadsX, NX1)),
                    static_cast<unsigned int>(calcNumBlocks(threadsY, NY)));
   sharedMemoryBytes = 8 * (threadsX + 1) * sizeof(float);
   uint2 tmpUint2;
   tmpUint2.x = numParticles;
   tmpUint2.y = 0;
#ifndef NO_THRUST
   DevMem<uint2, ParticleAllocator> topRowBlockBoundaries(numBlocks.x * numBlocks.y, tmpUint2);
   DevMem<uint2, ParticleAllocator> bottomRowBlockBoundaries(numBlocks.x * numBlocks.y, tmpUint2);
#else
   DevMem<uint2, ParticleAllocator> topRowBlockBoundaries(numBlocks.x * numBlocks.y);
   DevMem<uint2, ParticleAllocator> bottomRowBlockBoundaries(numBlocks.x * numBlocks.y);
   setDeviceArray(topRowBlockBoundaries.getPtr(), topRowBlockBoundaries.size(), tmpUint2);
   setDeviceArray(bottomRowBlockBoundaries.getPtr(), bottomRowBlockBoundaries.size(), tmpUint2);
#endif
   hipStreamSynchronize(stream);
   checkForCudaError("Finished prep for densGridPoints");
   densGridPoints<<<numBlocks, blockSize, sharedMemoryBytes, stream>>>(
      dev_rho.getPtr(),
      NX1,
      NY,
      dev_a1Sum.getPtr(),
      dev_a2Sum.getPtr(),
      dev_a3Sum.getPtr(),
      dev_a4Sum.getPtr(),
      cold,
      numParticles,
      NIJ
      );
   topRowBlockBoundaries.freeMem();
   bottomRowBlockBoundaries.freeMem();
   checkForCudaError("densGridPoints failed");

}

/******************************************************************************
  Function: dens
  Purpose: Find the charge density at grid points
  Parameters:
  -------------------
  dev_rho - The charge density at each grid point from 0 <= x < NX and 
            0 <= y < NY. rho is defined as rhoi - rhoe
  dev_rhoe - The charge density of the electrons at each grid point from 
             0 <= x < NX and 0 <= y < NY
  dev_rhoi - The charge density of the ions at each grid point from 
             0 <= x < NX and 0 <= y < NY
  d_eleHotLoc  - An array of the hot electron locations
  d_eleHotVel  - An array of the hot electron velocities
  d_eleColdLoc - An array of the cold electron locations
  d_eleColdVel - An array of the cold electron velocities
  d_ionHotLoc  - An array of the hot ion locations
  d_ionHotVel  - An array of the hot ion velocities
  d_ionColdLoc - An array of the cold ion locations
  d_ionColdVel - An array of the cold ion velocities
  numHotElectrons - The number of particles in dev_eleHot
  numColdElectrons - The number of particles in dev_eleCold
  numHotIons - The number of particles in dev_ionHot
  numColdIons - The number of particles in dev_ionCold
  sortEleHot - Sort the hot electron array
  sortEleCold - Sort the cold electron array
  sortIonHot - Sort the hot ion array
  sortIonCold - Sort the cold ion array
  stream - The hipStream_t to do all processing on
******************************************************************************/
void dens(DevMemF &dev_rho,
          DevMemF &dev_rhoe,
          DevMemF &dev_rhoi,
          DevMem<float2>& d_eleHotLoc, DevMem<float3>& d_eleHotVel,
          DevMem<float2>& d_eleColdLoc, DevMem<float3>& d_eleColdVel,
          DevMem<float2>& d_ionHotLoc, DevMem<float3>& d_ionHotVel,
          DevMem<float2>& d_ionColdLoc, DevMem<float3>& d_ionColdVel,
          unsigned int& numHotElectrons, unsigned int& numColdElectrons,
          unsigned int& numHotIons, unsigned int& numColdIons,
          bool sortEleHot, bool sortEleCold,
          bool sortIonHot, bool sortIonCold,
          DevStream &stream)
{
   dim3 *numBlocks;
   dim3 *blockSize;
   int threadsInBlock;

   // Clear Arrays
   dev_rho.zeroMem();
   dev_rhoe.zeroMem();
   dev_rhoi.zeroMem();

   // Calculate the rho from the hot electrons
   calcIntermediateRho(dev_rhoe, d_eleHotLoc, d_eleHotVel,
      numHotElectrons, false, true, sortEleHot, *stream);
   // Calculate the rho from the cold electrons
   calcIntermediateRho(dev_rhoe, d_eleColdLoc, d_eleColdVel,
      numColdElectrons, true, true, sortEleCold, *stream);
   // Calculate the rho from the hot ions
   calcIntermediateRho(dev_rhoi, d_ionHotLoc, d_ionHotVel,
      numHotIons, false, false, sortIonHot, *stream);
   // Calculate the rho from the cold ions
   calcIntermediateRho(dev_rhoi, d_ionColdLoc, d_ionColdVel,
      numColdIons, true, false, sortIonCold, *stream);

   // Double the rho at the top and bottom of the grid
   threadsInBlock = MAX_THREADS_PER_BLOCK;
   blockSize = new dim3(threadsInBlock);
   numBlocks = new dim3(static_cast<unsigned int>(calcNumBlocks(threadsInBlock, NX)));
   stream.synchronize();
   checkForCudaError("Before fixRhoGridTopBottom");
   fixRhoGridTopBottom<<<*numBlocks, *blockSize, 0, *stream>>>(
      dev_rhoe.getPtr(),
      dev_rhoi.getPtr(),
      NX1, NY);
   delete blockSize;
   delete numBlocks;
   checkForCudaError("fixRhoGridTopBottom");

   stream.synchronize();
   checkForCudaError("Before rhoi - rhoe");
   // Set rho = rhoi - rhoe
   subVector(dev_rhoi.getPtr(), dev_rhoe.getPtr(), 
             dev_rho.getPtr(), static_cast<unsigned int>(dev_rhoi.size()));
}
