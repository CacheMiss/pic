#include "hip/hip_runtime.h"
//*******************************************************************
// serial version of 2-d pic code 
// last modified: 7/21/2004
// comments: this version reflects changes made by  chakravaritty 
//           deverapalli in the dens subroutine (alterations to
//           rho() calcuation process)
//*******************************************************************
// cpotent2 k2----///nx-not-nx1///oct 1 02
//*******************************************************************
//       using one-dimensional fft and sm(k)
//        theta=0.05 radian bm=0.3
//       bzm = bm*cos(theta) ; bym = bm*sin(theta)   dx=dy=1  ; 
//       o: 9  oxygen ions have no  drift; delt=0.1
//       regular position distribution,   mass ratio =400  bx=1.2
//*******************************************************************
#include <assert.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <time.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <driver_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "array2d.h"
#include "commandline_options.h"
#include "dens.h"
#include "dev_mem_reuse.h"
#include "device_stats.h"
#include "device_utils.h"
#include "field.h"
#include "global_variables.h"
#include "inject.h"
#include "logging_thread.h"
#include "logging_types.h"
#include "movep.h"
#include "particle_allocator.h"
#include "pic_utils.h"
#include "potent2.h"
#include "precisiontimer.h"
#include "simulation_state.h"
#include "typedefs.h"

#ifdef _DEBUG
#ifndef DEBUG_TRACE
#define DEBUG_TRACE
#endif
#endif

void printFreeMem()
{
   DeviceStats &device(DeviceStats::getRef());

   std::cout << device.getFreeMemMb() << " Mb ("
             << static_cast<int>(device.getPercentFreeMem() * 100)
             << "%) of device memory is free." << std::endl;
}

void executePic(int argc, char *argv[])
{
   // Create output directory if necessary
   createOutputDir("run_output");

   // Clear the old error log
   FILE *errorLog = fopen(errorLogName.c_str(), "w");
   fclose(errorLog);

   int lfint = 5; // number of info intervals between output files

   CommandlineOptions &options(CommandlineOptions::getRef());
   options.parseArguments(argc, argv);
   lfint = options.getLogInterval();

   // Init Device
   DeviceStats &ref(DeviceStats::getRef());
   printFreeMem();

   DevMemReuse &reuseAllocator(DevMemReuse::getRef());
   reuseAllocator.setSizeX(NX1);
   reuseAllocator.setSizeY(NY);

   PrecisionTimer iterationTimer;
   //PrecisionTimer injectTimer;
   //PrecisionTimer densTimer;
   //PrecisionTimer potent2Timer;
   //PrecisionTimer fieldTimer;
   //PrecisionTimer movepTimer;
   LoggingThread &logger(LoggingThread::getRef());
   
   time_t startTime = time(0);
   time_t stopTime;
   SimulationState &simState(SimulationState::getRef());
   float maxSimTime = options.getMaxSimTime();
   int ind;
   int lfdint;
   int lfd;
   unsigned int nit;

   const std::size_t initialAllocSize = 5000000;
   const int neededParticles = NIJ*NX1; // Need this many particles in each array
   // 6 rands for hot electrons
   // 6 rands for cold electrons
   // 6 rands for hot ions
   // 6 rands for cold ions
   const int neededRands = neededParticles * 4 * 6;

#ifdef DEBUG_TRACE
   std::cout << "Initializing main storage..." << std::endl;
#endif
   // CUDA Variables
   int sharedMemoryBytes;
   // Device Memory Pointers
   DevMem<float2> d_eleHotLoc(initialAllocSize);
   DevMem<float3> d_eleHotVel(initialAllocSize);
   DevMem<float2> d_eleColdLoc(initialAllocSize);
   DevMem<float3> d_eleColdVel(initialAllocSize);
   DevMem<float2> d_ionHotLoc(initialAllocSize);
   DevMem<float3> d_ionHotVel(initialAllocSize);
   DevMem<float2> d_ionColdLoc(initialAllocSize);
   DevMem<float3> d_ionColdVel(initialAllocSize);
   DevMemF dev_phi(NY * NX1);
   DevMemF dev_ex((NY+1) * NX1); // An extra row is added to pad with zeros
   dev_ex.zeroMem();
   DevMemF dev_ey((NY+1) * NX1); // An extra row is added to pad with zeros
   dev_ey.zeroMem();
   DevMemF dev_rho(NX1 * NY);
   DevMemF dev_rhoe(NX1 * NY);
   DevMemF dev_rhoi(NX1 * NY);
   //DevMemF dev_xx(X_GRD);
   //DevMemF dev_yy(Y_GRD);
   DevMemF dev_randTable(neededRands);
   // End Device Memory Pointers

#ifdef DEBUG_TRACE
   std::cout << "Finished main storage" << std::endl;
#endif

   int percentComplete = 0; // Used to display progress to the user
   int percentSize = 0;

#ifdef DEBUG_TRACE
   std::cout << "Initializing random number generator" << std::endl;
#endif

   // Set up the random number generator
   hiprandGenerator_t randGenerator;
   hiprandCreateGenerator (&randGenerator, HIPRAND_RNG_PSEUDO_MTGP32);
   hiprandSetPseudoRandomGeneratorSeed(randGenerator, ISEED);

   nit = static_cast<int>((maxSimTime-TSTART)/DELT + 1); // determine number of iterations

   percentSize = nit / 100;

   simState.simTime = TSTART;
   lfd=LF-1;
   ind=0;
   lfdint=0;

   if(options.getRestartPoint() > 0)
   {
#ifdef DEBUG_TRACE
      std::cout << "Loading previous run data..." << std::endl;
#endif
      loadPrevSimState(options.getRestartPoint(), options.getRestartDir(),
         d_eleHotLoc, d_eleHotVel, d_eleColdLoc, d_eleColdVel,
         d_ionHotLoc, d_ionHotVel, d_ionColdLoc, d_ionColdVel,
         simState.numEleHot, simState.numEleCold,
         simState.numIonHot, simState.numIonCold);
      printf("INFO: Loaded %d hot electrons\n", simState.numEleHot);
      printf("INFO: Loaded %d cold electrons\n", simState.numEleCold);
      printf("INFO: Loaded %d hot ions\n", simState.numIonHot);
      printf("INFO: Loaded %d cold ions\n", simState.numIonCold);
      simState.iterationNum++;
      simState.simTime += DELT;
      lfd = 0;
      lfdint = 0;
      ind = simState.iterationNum / lfint + 1;
#ifdef DEBUG_TRACE
      std::cout << "previous run data loaded" << std::endl;
#endif
   }
   else
   {
      simState.iterationNum = 0;
   }
   // DEBUG
   //   {
   //      Array2dF *eleHot = new Array2dF(simState.numEleHot, 5);
   //      Array2dF *eleCold = new Array2dF(simState.numEleCold, 5);
   //      Array2dF *ionHot = new Array2dF(simState.numIonHot, 5);
   //      Array2dF *ionCold = new Array2dF(simState.numIonCold, 5);
   //      eleHot->loadRows(dev_eleHot, simState.numEleHot);
   //      eleCold->loadRows(dev_eleCold, simState.numEleCold);
   //      ionHot->loadRows(dev_ionHot, simState.numIonHot);
   //      ionCold->loadRows(dev_ionCold, simState.numIonCold);
   //      logger.logParticlesBinary(ind, eleHot, eleCold, ionHot, ionCold,
   //         simState.numEleHot, simState.numEleCold,
   //         simState.numIonHot, simState.numIonCold);
   //   }
   //   logger.flush();
   // END DEBUG

   std::cout << "Free mem after initial allocations:" << std::endl;
   printFreeMem();

   printf("nit=%d\n",nit);
   for (;simState.iterationNum<nit; simState.iterationNum++) 
   {
      if(percentComplete < 100 &&
         static_cast<int>(simState.iterationNum / percentSize) > percentComplete)
      {
         percentComplete = (int) simState.iterationNum / percentSize;
         printf("%d%% Complete\n", percentComplete);
         printFreeMem();
      }

      iterationTimer.start();

      simState.simTime +=DELT;
      lfd++;

      // Make sure I'm not out of memory
      const std::size_t ALLOC_INCREMENT = 1000000;
      if(simState.numEleHot + neededParticles > d_eleHotLoc.size())
      {
         std::cout << "Adding storage for hot electrons." << std::endl;
         // Scope these to keep the total required memory size lower
         {
            HostMem<float2> pos;
            pos = d_eleHotLoc;
            d_eleHotLoc.resize(d_eleHotLoc.size() + ALLOC_INCREMENT);
            d_eleHotLoc.copyArrayToDev(pos);
         }
         {
            HostMem<float3> vel;
            vel = d_eleHotVel;
            d_eleHotVel.resize(d_eleHotVel.size() + ALLOC_INCREMENT);
            d_eleHotVel.copyArrayToDev(vel);
         }
      }
      if(simState.numEleCold + neededParticles > d_eleColdLoc.size())
      {
         std::cout << "Adding storage for cold electrons." << std::endl;
         // Scope these to keep the total required memory size lower
         {
            HostMem<float2> pos;
            pos = d_eleColdLoc;
            d_eleColdLoc.resize(d_eleColdLoc.size() + ALLOC_INCREMENT);
            d_eleColdLoc.copyArrayToDev(pos);
         }
         {
            HostMem<float3> vel;
            vel = d_eleColdVel;
            d_eleColdVel.resize(d_eleColdVel.size() + ALLOC_INCREMENT);
            d_eleColdVel.copyArrayToDev(vel);
         }
      }
      if(simState.numIonHot + neededParticles > d_ionHotLoc.size())
      {
         std::cout << "Adding storage for hot ions." << std::endl;
         // Scope these to keep the total required memory size lower
         {
            HostMem<float2> pos;
            pos = d_ionHotLoc;
            d_ionHotLoc.resize(d_ionHotLoc.size() + ALLOC_INCREMENT);
            d_ionHotLoc.copyArrayToDev(pos);
         }
         {
            HostMem<float3> vel;
            vel = d_ionHotVel;
            d_ionHotVel.resize(d_ionHotVel.size() + ALLOC_INCREMENT);
            d_ionHotVel.copyArrayToDev(vel);
         }
      }
      if(simState.numIonCold + neededParticles > d_ionColdLoc.size())
      {
         std::cout << "Adding storage for cold ions." << std::endl;
         // Scope these to keep the total required memory size lower
         {
            HostMem<float2> pos;
            pos = d_ionColdLoc;
            d_ionColdLoc.resize(d_ionColdLoc.size() + ALLOC_INCREMENT);
            d_ionColdLoc.copyArrayToDev(pos);
         }
         {
            HostMem<float3> vel;
            vel = d_ionColdVel;
            d_ionColdVel.resize(d_ionColdVel.size() + ALLOC_INCREMENT);
            d_ionColdVel.copyArrayToDev(vel);
         }
      }

#ifdef DEBUG_TRACE
      std::cout << "Inject" << std::endl;
#endif

      // Prepare to call Inject
      // Generate the random numbers inject will need
      hiprandGenerateUniform(randGenerator, dev_randTable.getPtr(), neededRands);

      //injectTimer.start();
      const int injectThreadsPerBlock = MAX_THREADS_PER_BLOCK;
      dim3 injectNumBlocks(calcNumBlocks(injectThreadsPerBlock, neededParticles));
      dim3 injectBlockSize(injectThreadsPerBlock);
      sharedMemoryBytes = sizeof(float) * 5 * injectThreadsPerBlock;
      hipDeviceSynchronize();
      checkForCudaError("RandomGPU");
      // randomly inject new particles in top and bottom 
      inject<<<injectNumBlocks, injectBlockSize, sharedMemoryBytes>>>(
         d_eleHotLoc.getPtr(), d_eleHotVel.getPtr(), 
         d_eleColdLoc.getPtr(), d_eleColdVel.getPtr(), 
         d_ionHotLoc.getPtr(), d_ionHotVel.getPtr(), 
         d_ionColdLoc.getPtr(), d_ionColdVel.getPtr(), 
         DX, DY,
         simState.numEleHot, simState.numEleCold, 
         simState.numIonHot, simState.numIonCold,
         dev_randTable.getPtr(),
         dev_randTable.size(),
         NX1, NY1, NIJ
         );
      checkForCudaError("Inject failed");

      simState.numEleHot += neededParticles;
      simState.numEleCold += neededParticles;
      simState.numIonHot += neededParticles;
      simState.numIonCold += neededParticles;
      //hipDeviceSynchronize();
      //injectTimer.stop();

      // DEBUG
      //hipDeviceSynchronize();
      //logger.pushLogItem(
      //   new LogParticlesAscii(ind, d_eleHotLoc, d_eleHotVel,
      //   d_eleColdLoc, d_eleColdVel,
      //   d_ionHotLoc, d_ionHotVel,
      //   d_ionColdLoc, d_ionColdVel,
      //   simState.numEleHot, simState.numEleCold,
      //   simState.numIonHot, simState.numIonCold));
      //logger.flush();
      // END DEBUG

#ifdef DEBUG_TRACE
      std::cout << "Dens" << std::endl;
#endif
      //densTimer.start();
      // determine the charge density at the grid points
      dens(dev_rho, dev_rhoe,dev_rhoi, 
           d_eleHotLoc, d_eleColdLoc,
           d_ionHotLoc, d_ionColdLoc,
           simState.numEleHot, simState.numEleCold, 
           simState.numIonHot, simState.numIonCold);
      //hipDeviceSynchronize();
      //densTimer.stop();

      // Start DEBUG
      //hipDeviceSynchronize();
      //Array2dF *rho = new Array2dF(NY, NX1);
      //Array2dF *rhoe = new Array2dF(NY, NX1);
      //Array2dF *rhoi = new Array2dF(NY, NX1);
      //*rho = dev_rho;
      //*rhoe = dev_rhoe;
      //*rhoi = dev_rhoi;
      //logger.logRhoAscii(ind, rho, rhoe, rhoi);
      //logger.pushLogItem(
      //   new LogParticlesAscii(ind, d_eleHotLoc, d_eleHotVel,
      //   d_eleColdLoc, d_eleColdVel,
      //   d_ionHotLoc, d_ionHotVel,
      //   d_ionColdLoc, d_ionColdVel,
      //   simState.numEleHot, simState.numEleCold,
      //   simState.numIonHot, simState.numIonCold));
      //logger.flush();
      // End DEBUG

      //potent2Timer.start();
      // calculate potential at Grid points
      potent2(dev_phi, dev_rho);
      //hipDeviceSynchronize();
      //potent2Timer.stop();

#ifdef DEBUG_TRACE
      std::cout << "Field" << std::endl;
#endif
      //fieldTimer.start();
      // calculate E field at Grid points
      field(dev_ex,dev_ey,dev_phi);
      //hipDeviceSynchronize();
      //fieldTimer.stop();

      // DEBUG
      // hipDeviceSynchronize();
      // logger.pushLogItem(
      //    new LogParticlesAscii(ind, d_eleHotLoc, d_eleHotVel,
      //    d_eleColdLoc, d_eleColdVel,
      //    d_ionHotLoc, d_ionHotVel,
      //    d_ionColdLoc, d_ionColdVel,
      //    simState.numEleHot, simState.numEleCold,
      //    simState.numIonHot, simState.numIonCold));
      // logger.flush();
      // END DEBUG

      //movepTimer.start();
      // move ions
      hipStream_t movepStreams[4];
      for(int streamIdx = 0; streamIdx < 4; streamIdx++)
      {
         hipStreamCreate(&movepStreams[streamIdx]);
      }
      hipDeviceSynchronize();
#ifdef DEBUG_TRACE
      std::cout << "MoveHi" << std::endl;
#endif
      movep(d_ionHotLoc, d_ionHotVel, simState.numIonHot, 
         RATO, dev_ex, dev_ey, movepStreams[0]);
#ifdef DEBUG_TRACE
      std::cout << "MoveCi" << std::endl;
#endif
      movep(d_ionColdLoc, d_ionColdVel, simState.numIonCold, 
         RATO, dev_ex, dev_ey, movepStreams[1]);

      // move electrons
#ifdef DEBUG_TRACE
      std::cout << "MoveHe" << std::endl;
#endif
      movep(d_eleHotLoc, d_eleHotVel, simState.numEleHot, 
         (float) -1.0, dev_ex, dev_ey, movepStreams[2]);
#ifdef DEBUG_TRACE
      std::cout << "MoveCe" << std::endl;
#endif
      movep(d_eleColdLoc, d_eleColdVel, simState.numEleCold, 
         (float) -1.0, dev_ex, dev_ey, movepStreams[3]);
      for(int streamIdx = 0; streamIdx < 4; streamIdx++)
      {
         hipStreamDestroy(movepStreams[streamIdx]);
      }

      //hipDeviceSynchronize();
      //movepTimer.stop();

      // DEBUG
      //hipDeviceSynchronize();
      //logger.pushLogItem(
      //   new LogParticlesAscii(ind, d_eleHotLoc, d_eleHotVel,
      //   d_eleColdLoc, d_eleColdVel,
      //   d_ionHotLoc, d_ionHotVel,
      //   d_ionColdLoc, d_ionColdVel,
      //   simState.numEleHot, simState.numEleCold,
      //   simState.numIonHot, simState.numIonCold));
      //logger.flush();
      // END DEBUG

      iterationTimer.stop();

      if (lfd >= LF) 
      {
         hipDeviceSynchronize();
         logger.logInfo(ind, simState.simTime, 
            simState.numEleHot + simState.numEleCold,
            simState.numIonHot + simState.numIonCold);
         logger.logForPerformance(ind, simState.simTime, 
            simState.numEleHot, simState.numEleCold, 
            simState.numIonHot, simState.numIonCold, 
            (unsigned int) iterationTimer.intervalInMilliS(),
            0, 0, 0, 0, 0,
            options.getRestartPoint() > 0 ? true : false);
            //(unsigned int) injectTimer.intervalInMilliS(),
            //(unsigned int) densTimer.intervalInMilliS(),
            //(unsigned int) potent2Timer.intervalInMilliS(),
            //(unsigned int) fieldTimer.intervalInMilliS(),
            //(unsigned int) movepTimer.intervalInMilliS());
         lfdint = lfdint + 1;
         if (lfdint >= lfint) 
         {
            Array2dF *phi = new Array2dF(NY, NX1);
            Array2dF *ex = new Array2dF(NY+1, NX1);
            Array2dF *ey = new Array2dF(NY+1, NX1);
            Array2dF *rho = new Array2dF(NY, NX1);
            Array2dF *rhoe = new Array2dF(NY, NX1);
            Array2dF *rhoi = new Array2dF(NY, NX1);

            // Move computations back to host
            *rho = dev_rho;
            *rhoe = dev_rhoe;
            *rhoi = dev_rhoi;
            *phi = dev_phi;
      
            if(options.getLogInAscii())
            {
               logger.pushLogItem(
                  new LogParticlesAscii(ind, d_eleHotLoc, d_eleHotVel,
                  d_eleColdLoc, d_eleColdVel,
                  d_ionHotLoc, d_ionHotVel,
                  d_ionColdLoc, d_ionColdVel,
                  simState.numEleHot, simState.numEleCold,
                  simState.numIonHot, simState.numIonCold));
               logger.logRhoAscii(ind, rho, rhoe, rhoi);
               logger.logPhiAscii(ind, phi);
            }
            else
            {
               logger.pushLogItem(
                  new LogParticlesBinary(ind, d_eleHotLoc, d_eleHotVel,
                  d_eleColdLoc, d_eleColdVel,
                  d_ionHotLoc, d_ionHotVel,
                  d_ionColdLoc, d_ionColdVel,
                  simState.numEleHot, simState.numEleCold,
                  simState.numIonHot, simState.numIonCold));
               logger.logRhoBinary(ind, rho, rhoe, rhoi);
               logger.logPhiBinary(ind, phi);
            }
            lfdint = 0;
         }
         lfd=0 ;
         ind=ind+1;
      }
   }

   stopTime = time(0);
   unsigned int timeMin = (unsigned int)(stopTime - startTime) / 60;
   unsigned int timeSec = (unsigned int)(stopTime - startTime) % 60;

   std::string runStatisticsFn = outputDir + "/run_statistics.txt";
   FILE *f = fopen(runStatisticsFn.c_str(), "w");
   fprintf(f, "nit %u reached at %u min %u sec\n", nit, timeMin, timeSec);
   fclose(f);

}

int main(int argc, char *argv[])
{
   try
   {
      DeviceStats &ref(DeviceStats::getRef());
   }
   catch(CudaRuntimeError e)
   {
      std::cout << e.what() << std::endl;
      throw;
   }
   try
   {
      executePic(argc, argv);
   }
   catch(CudaRuntimeError e)
   {
      std::cout << e.what() << std::endl;
      ParticleAllocator::getRef().cleanup();
      DevMemReuse::getRef().cleanup();
      hipDeviceReset();
      throw;
   }

   ParticleAllocator::getRef().cleanup();
   DevMemReuse::getRef().cleanup();
   hipDeviceReset();

   return 0;
}
