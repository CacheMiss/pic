#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <fstream>
#include <iostream>
#include <thrust/sort.h>

#include "dev_mem.h"
#include "device_stats.h"
#include "precisiontimer.h"

#define NX1 512
#define NY1 10000

struct Particle_t
{
   float2 pos;
   float3 vel;
};

__host__ __device__
bool operator<(const Particle_t& lhs, const Particle_t& rhs)
{
   unsigned int lhsX = static_cast<unsigned int>(lhs.pos.x);
   unsigned int lhsY = static_cast<unsigned int>(lhs.pos.y);
   unsigned int rhsX = static_cast<unsigned int>(rhs.pos.x);
   unsigned int rhsY = static_cast<unsigned int>(rhs.pos.y);
   bool ret = false;
   if(lhsY < rhsY)
   {
      ret = true;
   }
   else if(lhsY == rhsY)
   {
      if(lhsX < rhsX)
      {
         ret = true;
      }
   }
   return ret;
}

__global__
void initParticles(float2* pos, float *randArray, unsigned int numParticles)
{
   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   if(threadX < numParticles)
   {
      pos[threadX].x = NX1 * randArray[threadX];
      pos[threadX].y = NY1 * randArray[numParticles+threadX];
   }
}

__global__
void initParticles(Particle_t* particle, float *randArray, unsigned int numParticles)
{
   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   if(threadX < numParticles)
   {
      particle[threadX].pos.x = NX1 * randArray[threadX];
      particle[threadX].pos.y = NY1 * randArray[numParticles+threadX];
   }
}

__global__
void binParticles(const float2* pos, unsigned int* bins, unsigned int numParticles)
{
   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   if(threadX < numParticles)
   {
      bins[threadX] = NX1 * pos[threadX].y + pos[threadX].x;
   }
}

void timeSorts(const unsigned int numParticles, float& timeBin, float& timeFull,
               hipEvent_t& eventBeg, hipEvent_t& eventEnd)
{
   const int neededRands = numParticles * 2;
   DevMem<float> randArray(neededRands);

   PrecisionTimer timer;

   hiprandGenerator_t randGenerator;
   hiprandCreateGenerator (&randGenerator, HIPRAND_RNG_PSEUDO_MTGP32);
   hiprandSetPseudoRandomGeneratorSeed(randGenerator, 1);
   hiprandGenerateUniform(randGenerator, randArray.getPtr(), neededRands);
   hiprandDestroyGenerator(randGenerator);

   DevMem<float2> pos(numParticles);
   DevMem<unsigned int> bins(numParticles);

   const unsigned int threadsPerBlock = 512;
   const unsigned int numBlocks = (numParticles + threadsPerBlock - 1) / threadsPerBlock;
   initParticles<<<numBlocks, threadsPerBlock>>>(pos.getPtr(), randArray.getPtr(), numParticles);
   checkCuda(hipGetLastError());
   binParticles<<<numBlocks, threadsPerBlock>>>(pos.getPtr(), bins.getPtr(), numParticles);
   checkCuda(hipGetLastError());
   checkCuda(hipDeviceSynchronize());

   checkCuda(hipEventRecord(eventBeg));
   thrust::sort_by_key(bins.getThrustPtr(), bins.getThrustPtr()+numParticles, pos.getThrustPtr());
   checkCuda(hipEventRecord(eventEnd));
   checkCuda(hipDeviceSynchronize());

   checkCuda(hipEventElapsedTime(&timeBin, eventBeg, eventEnd));

   pos.freeMem();
   bins.freeMem();

   DevMem<Particle_t> particles(numParticles);
   initParticles<<<numBlocks, threadsPerBlock>>>(particles.getPtr(), randArray.getPtr(), numParticles);
   checkCuda(hipDeviceSynchronize());

   checkCuda(hipEventRecord(eventBeg));
   thrust::sort(particles.getThrustPtr(), particles.getThrustPtr()+numParticles);
   checkCuda(hipEventRecord(eventEnd));
   checkCuda(hipDeviceSynchronize());

   checkCuda(hipEventElapsedTime(&timeFull, eventBeg, eventEnd));
}

int main()
{
   DeviceStats &device = DeviceStats::getRef();

   const int maxParticles = 5000000;

   hipEvent_t eventBeg;
   hipEvent_t eventEnd;

   checkCuda(hipEventCreate(&eventBeg));
   checkCuda(hipEventCreate(&eventEnd));

   float timeBin;
   float timeFull;

   std::ofstream sortTimes("sortTimes.txt");
   sortTimes << "numParticles,binSortTime(ms),fullSortTime(ms)" << std::endl;

   for(int i = 100000; i <= maxParticles; i+=100000)
   {
      timeSorts(i, timeBin, timeFull, eventBeg, eventEnd);
      std::cout << i << " particles; binSort: " << timeBin << "ms fullSort: " << timeFull << "ms" << std::endl;
      sortTimes << i << "," << timeBin << "," << timeFull << std::endl;
   }

   checkCuda(hipEventDestroy(eventBeg));
   checkCuda(hipEventDestroy(eventEnd));

   return 0;
}