#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <fstream>
#include <iostream>
#include <thrust/sort.h>

#include "dev_mem.h"
#include "device_stats.h"
#include "precisiontimer.h"

#define NX1 512
#define NY1 10000

struct Particle_t
{
   float2 pos;
   float3 vel;
};

__host__ __device__
bool operator<(const Particle_t& lhs, const Particle_t& rhs)
{
   if(lhs.pos.y < rhs.pos.y)
   {
      return true;
   }
   else if(lhs.pos.y == rhs.pos.y)
   {
      if(lhs.pos.x < rhs.pos.x)
      {
         return true;
      }
   }
   else
   {
      return false;
   }
   return false;
}

__global__
void initParticles(float2* pos, float *randArray, unsigned int numParticles)
{
   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   if(threadX < numParticles)
   {
      pos[threadX].x = NX1 * randArray[threadX];
      pos[threadX].y = NY1 * randArray[numParticles+threadX];
   }
}

__global__
void initParticles(Particle_t* particle, float *randArray, unsigned int numParticles)
{
   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   if(threadX < numParticles)
   {
      particle[threadX].pos.x = NX1 * randArray[threadX];
      particle[threadX].pos.y = NY1 * randArray[numParticles+threadX];
   }
}

__global__
void binParticles(const float2* pos, unsigned int* bins, unsigned int numParticles)
{
   unsigned int threadX = blockDim.x * blockIdx.x + threadIdx.x;
   if(threadX < numParticles)
   {
      bins[threadX] = NX1 * pos[threadX].y + pos[threadX].x;
   }
}

void timeSorts(const unsigned int numParticles, float& timeBin, float& timeFull,
               hipEvent_t& eventBeg, hipEvent_t& eventEnd)
{
   const int neededRands = numParticles * 2;
   DevMem<float> randArray(neededRands);

   PrecisionTimer timer;

   hiprandGenerator_t randGenerator;
   hiprandCreateGenerator (&randGenerator, HIPRAND_RNG_PSEUDO_MTGP32);
   hiprandSetPseudoRandomGeneratorSeed(randGenerator, 1);
   hiprandGenerateUniform(randGenerator, randArray.getPtr(), neededRands);
   hiprandDestroyGenerator(randGenerator);

   DevMem<float2> pos(numParticles);
   DevMem<unsigned int> bins(numParticles);

   const unsigned int threadsPerBlock = 512;
   const unsigned int numBlocks = (numParticles + threadsPerBlock - 1) / threadsPerBlock;
   initParticles<<<numBlocks, threadsPerBlock>>>(pos.getPtr(), randArray.getPtr(), numParticles);
   checkCuda(hipGetLastError());
   binParticles<<<numBlocks, threadsPerBlock>>>(pos.getPtr(), bins.getPtr(), numParticles);
   checkCuda(hipGetLastError());
   checkCuda(hipDeviceSynchronize());

   checkCuda(hipEventRecord(eventBeg));
   thrust::sort_by_key(bins.getThrustPtr(), bins.getThrustPtr()+numParticles, pos.getThrustPtr());
   checkCuda(hipEventRecord(eventEnd));
   checkCuda(hipDeviceSynchronize());

   checkCuda(hipEventElapsedTime(&timeBin, eventBeg, eventEnd));

   pos.freeMem();
   bins.freeMem();

   DevMem<Particle_t> particles(numParticles);
   initParticles<<<numBlocks, threadsPerBlock>>>(particles.getPtr(), randArray.getPtr(), numParticles);
   checkCuda(hipDeviceSynchronize());

   checkCuda(hipEventRecord(eventBeg));
   thrust::sort(particles.getThrustPtr(), particles.getThrustPtr()+numParticles);
   checkCuda(hipEventRecord(eventEnd));
   checkCuda(hipDeviceSynchronize());

   checkCuda(hipEventElapsedTime(&timeFull, eventBeg, eventEnd));
}

int main()
{
   DeviceStats &device = DeviceStats::getRef();

   const int maxParticles = 5000000;

   hipEvent_t eventBeg;
   hipEvent_t eventEnd;

   checkCuda(hipEventCreate(&eventBeg));
   checkCuda(hipEventCreate(&eventEnd));

   float timeBin;
   float timeFull;

   std::ofstream sortTimes("sortTimes.txt");
   sortTimes << "numParticles,binSortTime(ms),fullSortTime(ms)" << std::endl;

   for(int i = 100000; i <= maxParticles; i+=100000)
   {
      timeSorts(i, timeBin, timeFull, eventBeg, eventEnd);
      std::cout << i << " particles; binSort: " << timeBin << "ms fullSort: " << timeFull << "ms" << std::endl;
      sortTimes << i << "," << timeBin << "," << timeFull << std::endl;
   }

   checkCuda(hipEventDestroy(eventBeg));
   checkCuda(hipEventDestroy(eventEnd));

   return 0;
}